#include "hip/hip_runtime.h"
// k <= 6
#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <vector>

#include <hipcub/hipcub.hpp>
#include <omp.h>

#include "dataloader.h"
#include "graph.h"
#include "component/utils.cuh"

constexpr int BITS_PER_PARTITION = 64;
constexpr int LENGTH = 6;
// 单个 partition_num = ceil(size / 64)，表示多少个 uint 表示一个二进制集合

// int* partition_num;
int64_t* start_ptr;

constexpr int THREADS_PER_BLOCK = 128;


/*
__device__ int popcount(unsigned int* x, int* partition_num) {
  int result = 0;
  for (int i = 0; i < *partition_num; ++i) {
    result += __popc(x[i]);
  }
  return result;
}
*/

constexpr int MAX_THREAD_BLOCK = 0x0FFFFFFF;  // 好像可以设成 2^31-1？
constexpr int MAX_DEPTH = 8;
constexpr int THREADS_PER_WARP = 32;
constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_WARP;

__device__ unsigned long long dev_sum = 0;

/**
 * @note: 每一个 Warp 负责一棵搜索子树，Warp 里面的线程一起负责求交
 *        后面的版本是最开始写的暴力：线程块内的每一个线程独立处理一棵搜索子树
 * @todo: 1. Warp 细化（有时候 32 不是最优）
 *        2. 感觉可以加启发式（类似木桶理论），而不是严格按照 vertex 编号顺序来
 */
__global__ void traverse_on_warp_partition(int* n, int* thread_block_num,
                                           unsigned long long* binary_adj,
                                           int64_t* vertex, int64_t* start_ptr,
                                           int* k) {
  __shared__ unsigned long long cache[WARPS_PER_BLOCK];
  // __shared__ int stack_vertex_pool[WARPS_PER_BLOCK * MAX_DEPTH];
  // __shared__ int* stack_vertex_ptr[WARPS_PER_BLOCK];
  __shared__ unsigned long long stack_binary_adj_pool[THREADS_PER_BLOCK * MAX_DEPTH];
  __shared__ unsigned long long* stack_binary_adj_ptr[WARPS_PER_BLOCK];
  int stack_vertex[MAX_DEPTH];
  // int stack_binary_adj[MAX_DEPTH * THREADS_PER_WARP];

  if (threadIdx.x == 0) {
    for (int i = 0; i < WARPS_PER_BLOCK; ++i) {
      cache[i] = 0;
    }
    for (int i = 0; i < WARPS_PER_BLOCK; ++i) {
      // stack_vertex_ptr[i] = stack_vertex_pool + i * MAX_DEPTH;
      stack_binary_adj_ptr[i] =
          stack_binary_adj_pool + i * THREADS_PER_WARP * MAX_DEPTH;
    }
  }
  __syncthreads();

  // int u = blockIdx.x;
  int warpidx = threadIdx.x / THREADS_PER_WARP;
  // int size = (int)(vertex[u + 1] - vertex[u]);
  bool is_main_thread = (threadIdx.x % THREADS_PER_WARP == 0);
  // int* stack_vertex = stack_vertex_ptr[warpidx];
  unsigned long long* stack_binary_adj = stack_binary_adj_ptr[warpidx];

  for (int64_t o = (int64_t)blockIdx.x * WARPS_PER_BLOCK + warpidx;
       o < vertex[*n]; o += (int64_t)thread_block_num * WARPS_PER_BLOCK) {
    int l = 0, r = *n - 1;
    while (l != r) {
      int middle = (l + r >> 1) + 1;
      if (vertex[middle] <= o) {
        l = middle;
      } else {
        r = middle - 1;
      }
    }
    // u 是结点编号，v 实际上是在 adj(u) 中的相对编号
    int u = l;
    int v = (int)(o - vertex[u]);
    int size = (int)(vertex[u + 1] - vertex[u]);
    // ceil(size / 64)
    int partition_num = (size - 1) / BITS_PER_PARTITION + 1;

    int top = 0;
    stack_vertex[top] = v;
    stack_vertex[top + 1] = -1;
    if (is_main_thread) {
      for (int i = 0; i < partition_num; ++i) {
        stack_binary_adj[i] = binary_adj[start_ptr[u] + v * partition_num + i];
      }
    }
    // __syncwarp();

    unsigned long long sum = 0;
    // 预定义 实际上是用于迭代的。好像会快一些？
    int partitionidx;
    int bitidx;
    bool ok;
    int foobar;

    // 因为 partition_num 的定义后面改过，所以下面注释里的一些地方可能有问题
    while (~top) {
      if (top + 3 == *k) {
        if (is_main_thread) {
          // sum += popcount(stack_binary_adj + (*partition_num) * top,
          // partition_num);
          foobar = partition_num * top;
          for (int i = 0; i < partition_num; ++i) {
            sum += __popcll(stack_binary_adj[foobar + i]);
          }
        }
        --top;
      } else {
        ++stack_vertex[top + 1];

        partitionidx = stack_vertex[top + 1] >> LENGTH;                   // / 64
        bitidx = stack_vertex[top + 1] & (BITS_PER_PARTITION - 1);        // % 64
        ok = false;

        while (stack_vertex[top + 1] < size) {
          // 小优化：如果后面都没有 1 位了，直接跳过（最多）32 个点
          // 不过实际效果好像并不是很显著？
          // if (!(stack_binary_adj[top * (*partition_num) + partitionidx] >>
          // bitidx)) {
          if (!(binary_adj[start_ptr[u] + stack_vertex[top] * partition_num +
                           partitionidx] >>
                bitidx)) {
            stack_vertex[top + 1] += BITS_PER_PARTITION - bitidx;
            bitidx = 0;
            ++partitionidx;
            continue;
          }
          // if ((stack_binary_adj[top * (*partition_num) + partitionidx] >>
          // bitidx) & 1) {
          if ((binary_adj[start_ptr[u] + stack_vertex[top] * partition_num +
                          partitionidx] >>
               bitidx) &
              1) {
            ++top;
            int i = threadIdx.x % THREADS_PER_WARP;
            if (i < partition_num) {
              stack_binary_adj[top * partition_num + i] =
                  stack_binary_adj[(top - 1) * partition_num + i] &
                  binary_adj[start_ptr[u] + stack_vertex[top] * partition_num +
                             i];
            }
            // __syncwarp();

            ok = true;
            stack_vertex[top + 1] = -1;
            break;
          }
          ++stack_vertex[top + 1];
          if (++bitidx == BITS_PER_PARTITION) {
            bitidx = 0;
            ++partitionidx;
          }
        }

        if (!ok) {
          --top;
        }
      }
    }

    if (is_main_thread) {
      cache[warpidx] += sum;
    }
  }
  __syncthreads();

  if (threadIdx.x == 0) {
    for (int i = 1; i < WARPS_PER_BLOCK; ++i) {
      cache[0] += cache[i];
    }
    atomicAdd(&dev_sum, cache[0]);
  }
}




template<int depth>
__device__ void traverse_func(unsigned long long & sum, unsigned long long * stack_binary_adj, unsigned long long* binary_adj, int64_t * start_ptr, int k, int u, int size, int partition_num){
  if(depth + 3 == k) {
    int foobar = partition_num * depth;
    for (int i = 0; i < partition_num; ++i) {
      sum += __popcll(stack_binary_adj[foobar + i]);
    }
  } else {
    for(int t = 0; t < size; t++) {
      int partitionidx = t >> LENGTH;
      int bitidx = t & (BITS_PER_PARTITION - 1);
      if (!(stack_binary_adj[depth * partition_num + partitionidx] >> bitidx)) {
        t += BITS_PER_PARTITION - bitidx - 1;
        continue;
      }
      if ((stack_binary_adj[depth * partition_num + partitionidx] >> bitidx) & 1) {
            for (int i = 0; i < partition_num; ++i) {
              stack_binary_adj[(depth + 1) * partition_num + i] =
                  stack_binary_adj[(depth) * partition_num + i] &
                  binary_adj[start_ptr[u] + t * partition_num + i];
            }
        traverse_func<depth + 1>(sum, stack_binary_adj, binary_adj, start_ptr, k, u, size, partition_num);
      }
    }
  }
}

template<>
__device__ void traverse_func<MAX_DEPTH>(unsigned long long & sum, unsigned long long * stack_binary_adj, unsigned long long* binary_adj, int64_t * start_ptr, int k, int u, int size, int partition_num) {
  assert(false);
}

__global__ void traverse(unsigned long long* binary_adj, int64_t* vertex,
                         int64_t* start_ptr, int* k) {

  typedef hipcub::BlockReduce<unsigned long long, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int u = blockIdx.x;
  int tid = threadIdx.x;
  int size = (int)(vertex[u + 1] - vertex[u]);
  int partition_num = (size - 1) / BITS_PER_PARTITION + 1;

  int stack_vertex[MAX_DEPTH];
  unsigned long long stack_binary_adj[MAX_DEPTH * 20];

  unsigned long long sum = 0;

  for(int v = tid; v < size; v += THREADS_PER_BLOCK) {
    int top = 0;

    stack_vertex[top] = v;
    stack_vertex[top + 1] = -1;
    for (int i = 0; i < partition_num; ++i) {
      stack_binary_adj[i] = binary_adj[start_ptr[u] + v * partition_num + i];
    }


    int partitionidx;
    int bitidx;
    bool ok;
    int foobar;

    while (~top) {
      if (top + 3 == *k) {
        // sum += popcount(stack_binary_adj + (*partition_num) * top,
        // partition_num);
        foobar = partition_num * top;
        for (int i = 0; i < partition_num; ++i) {
          sum += __popcll(stack_binary_adj[foobar + i]);
        }
        --top;
      } else {
        ++stack_vertex[top + 1];

        partitionidx = stack_vertex[top + 1] >> LENGTH;
        bitidx = stack_vertex[top + 1] & (BITS_PER_PARTITION - 1);
        ok = false;

        while (stack_vertex[top + 1] < size) {
          if (!(stack_binary_adj[top * partition_num + partitionidx] >>
                bitidx)) {
            stack_vertex[top + 1] += BITS_PER_PARTITION - bitidx;
            bitidx = 0;
            ++partitionidx;
            continue;
          }
          if ((stack_binary_adj[top * partition_num + partitionidx] >> bitidx) &
              1) {
            ++top;
            for (int i = 0; i < partition_num; ++i) {
              stack_binary_adj[top * partition_num + i] =
                  stack_binary_adj[(top - 1) * partition_num + i] &
                  binary_adj[start_ptr[u] + stack_vertex[top] * partition_num +
                             i];
            }
            ok = true;
            stack_vertex[top + 1] = -1;
            break;
          }
          ++stack_vertex[top + 1];
          if (++bitidx == BITS_PER_PARTITION) {
            bitidx = 0;
            ++partitionidx;
          }
        }

        if (!ok) {
          --top;
        }
      }
    }


  }

  __syncthreads();
  
  unsigned long long aggregate = BlockReduce(temp_storage).Sum(sum);

  if(tid == 0) {
    atomicAdd(&dev_sum, aggregate);
  }

  // delete[] stack_vertex;
  // delete[] stack_binary_adj;
}


__global__ void traverse_recursive(unsigned long long* binary_adj, int64_t* vertex,
                         int64_t* start_ptr, int* k) {

  typedef hipcub::BlockReduce<unsigned long long, THREADS_PER_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;

  int u = blockIdx.x;
  int tid = threadIdx.x;
  int size = (int)(vertex[u + 1] - vertex[u]);
  int partition_num = (size - 1) / BITS_PER_PARTITION + 1;


  unsigned long long stack_binary_adj[MAX_DEPTH * 20];

  unsigned long long sum = 0;

  for(int v = tid; v < size; v += THREADS_PER_BLOCK) {
    for (int i = 0; i < partition_num; ++i) {
      stack_binary_adj[i] = binary_adj[start_ptr[u] + v * partition_num + i];
    }
    traverse_func<0>(sum, stack_binary_adj, binary_adj, start_ptr, *k, u, size, partition_num);
  }

  __syncthreads();

  unsigned long long aggregate = BlockReduce(temp_storage).Sum(sum);
  if(tid == 0) {
    atomicAdd(&dev_sum, aggregate);
  }

}

void k_clique_counting(Graph* g, int k) {
  int n = g->v_cnt;
  long long m = g->e_cnt;
  start_ptr = new int64_t[n + 1];

  start_ptr[0] = 0;
  for (int u = 0; u < n; ++u) {
    int size = g->vertex[u + 1] - g->vertex[u];
    start_ptr[u + 1] = start_ptr[u] + size * ((size - 1) / BITS_PER_PARTITION + 1);
  }

  unsigned long long* binary_adj = new unsigned long long [start_ptr[n]];
  for (long long i = start_ptr[n + 1] - 1; ~i; --i) {
    binary_adj[i] = 0;
  }

  #pragma omp parallel for num_threads(64) schedule(dynamic)
  for (int u = 0; u < n; ++u) {
    int size = (int)(g->vertex[u + 1] - g->vertex[u]);
    int partition_num = (size - 1) / BITS_PER_PARTITION + 1;
    for (int64_t i = g->vertex[u]; i < g->vertex[u + 1]; ++i) {
      int v = g->edge[i];
      int64_t p = g->vertex[u];
      int64_t q = g->vertex[v];
      int partitionidx = 0;
      int bitidx = 0;
      while (p < g->vertex[u + 1]) {
        while (q < g->vertex[v + 1] && g->edge[q] < g->edge[p]) {
          ++q;
        }
        if (q < g->vertex[v + 1] && g->edge[p] == g->edge[q]) {
          binary_adj[start_ptr[u] + partition_num * (i - g->vertex[u]) +
                     partitionidx] |= 1ull << bitidx;
        }
        ++p;
        if (++bitidx == BITS_PER_PARTITION) {
          bitidx = 0;
          ++partitionidx;
        }
      }
    }
  }

  // printf("%d %lld\n", g->v_cnt, g->e_cnt);
  // for (int i = 0; i <= g->v_cnt; ++i) {
  //   std::cerr << g->vertex[i] << " \n"[i == g->v_cnt];
  // }
  // for (long long i = 0; i < g->e_cnt; ++i) {
  //   std::cerr << binary_adj[i] << " \n"[i + 1 == g->e_cnt];
  // }

  unsigned long long* gpu_binary_adj;
  int64_t* gpu_vertex;
  int64_t* gpu_start_ptr;
  int* gpu_k;
  int* gpu_n;
  int* gpu_thread_block_num;
  int thread_block_num =
      (int)std::min((long long)MAX_THREAD_BLOCK,
                    (m * THREADS_PER_WARP - 1) / THREADS_PER_BLOCK + 1);

  // printf("start_ptr:%lld\n", start_ptr[n]);

  gpuErrchk( hipMalloc((void**)&gpu_binary_adj, start_ptr[n] * sizeof(unsigned long long)) );
  gpuErrchk( hipMalloc((void**)&gpu_vertex, (n + 1) * sizeof(int64_t)) );
  gpuErrchk( hipMalloc((void**)&gpu_start_ptr, (n + 1) * sizeof(int64_t)) );
  gpuErrchk( hipMalloc((void**)&gpu_k, sizeof(int)) );
  gpuErrchk( hipMalloc((void**)&gpu_n, sizeof(int)) );
  gpuErrchk( hipMalloc((void**)&gpu_thread_block_num, sizeof(int)) );

  gpuErrchk( hipMemcpy(gpu_binary_adj, binary_adj, start_ptr[n] * sizeof(unsigned long long),
             hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(gpu_vertex, g->vertex, (n + 1) * sizeof(int64_t),
             hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(gpu_start_ptr, start_ptr, (n + 1) * sizeof(int64_t),
             hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(gpu_k, &k, sizeof(int), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(gpu_n, &n, sizeof(int), hipMemcpyHostToDevice) );
  gpuErrchk( hipMemcpy(gpu_thread_block_num, &thread_block_num, sizeof(int),
             hipMemcpyHostToDevice) );

  clock_t start_time = clock();
  printf("Counting function is running...\n");

  unsigned long long sum = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(dev_sum), &sum, sizeof(unsigned long long));

  traverse_recursive<<<n, THREADS_PER_BLOCK>>>(gpu_binary_adj, gpu_vertex, gpu_start_ptr,
                                     gpu_k);
  // traverse_on_warp_partition<<<thread_block_num, THREADS_PER_BLOCK>>>
  //    (gpu_n, gpu_thread_block_num, gpu_binary_adj, gpu_vertex, gpu_start_ptr,
  //    gpu_k);

  gpuErrchk( hipPeekAtLastError() );
  gpuErrchk( hipDeviceSynchronize() );
  
  gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(unsigned long long)) );

  printf("Answer: %llu\n", sum);

  clock_t end_time = clock();
  printf("Counting time cost: %.6lf s.\n",
         (double)(end_time - start_time) / CLOCKS_PER_SEC);

  hipFree(gpu_binary_adj);
  hipFree(gpu_vertex);
  hipFree(gpu_start_ptr);
  hipFree(gpu_k);
  hipFree(gpu_n);
  hipFree(gpu_thread_block_num);
}

// test
/*
int main() {
  Graph *original_g = new Graph();
  Graph *g;
  int k;

  freopen("data/3.in", "r", stdin);
  scanf("%d%lld%d", &(original_g->v_cnt), &(original_g->e_cnt), &k);
  original_g->edge = (int*)malloc(original_g->e_cnt * sizeof(int));
  original_g->vertex = (int64_t*)malloc((original_g->v_cnt + 1) *
sizeof(int64_t)); for (long long i = 0; i < original_g->e_cnt; ++i) {
    scanf("%d", &original_g->edge[i]);
  }
  for (int i = 0; i <= original_g->v_cnt; ++i) {
    scanf("%lld", &original_g->vertex[i]);
  }

  orientation_init(original_g, g);
  // std::cerr << partition_num << '\n';

  // printf("%d %lld\n", g->v_cnt, g->e_cnt);
  // for (int i = 0; i <= g->v_cnt; ++i) {
  //   std::cerr << g->vertex[i] << " \n"[i == g->v_cnt];
  // }
  // for (long long i = 0; i < g->e_cnt; ++i) {
  //   std::cerr << g->edge[i] << " \n"[i + 1 == g->e_cnt];
  // }

  clock_t start_time = clock();

  if (k == 1) {
    printf("count %d\n", g->v_cnt);
  } else if (k == 2) {
    printf("count %lld\n", g->e_cnt);
  } else {
    k_clique_counting(g, k);
    hipDeviceSynchronize();
  }

  clock_t end_time = clock();
  printf("Time: %lfs.\n", (double)((end_time - start_time) / CLOCKS_PER_SEC));
  return 0;
}
*/

int main(int argc, char** argv) {
  Graph *original_g, *g;
  DataLoader D;

  if (argc != 3) {
    printf("usage: %s graph_file clique_size\n", argv[0]);
    return 0;
  }
  // 重要！这里得到的 original_g 里的边是有序的
  // 也就是 edge[vertex[u] ~ vertex[u+1]-1] 是有序数组
  // 后面的 g 亦是如此
  bool ok = D.fast_load(original_g, argv[1]);
  if (!ok) {
    printf("Load data failed.\n");
    return 0;
  }

  degree_orientation_init(original_g, g);
  // degeneracy_orientation_init(original_g, g);

  int k = atoi(argv[2]);
  if (k == 1) {
    printf("count %d\n", g->v_cnt);
  } else if (k == 2) {
    printf("count %lld\n", g->e_cnt);
  } else {
    k_clique_counting(g, k);
  }

  return 0;
}