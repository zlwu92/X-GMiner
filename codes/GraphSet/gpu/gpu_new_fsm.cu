#include "hip/hip_runtime.h"
// 第一种尝试，每个 labeled pattern 用一个 kernel
#define THRUST_IGNORE_CUB_VERSION_CHECK
#undef NDEBUG
#include <sys/time.h>

#include <cassert>
#include <cstring>
#include <cstdint>
#include <string>
#include <algorithm>

#include <hipcub/hipcub.hpp>

#include "graph.h"
#include "dataloader.h"
#include "vertex_set.h"
#include "common.h"
#include "schedule_IEP.h"
#include "timeinterval.h"

#include "component/utils.cuh"
#include "component/gpu_schedule.cuh"
#include "component/gpu_vertex_set.cuh"
#include "component/gpu_bitvector.cuh"
#include "component/gpu_const.cuh"


TimeInterval allTime;
TimeInterval tmpTime;


int get_pattern_edge_num(const Pattern& p)
{
    int edge_num = 0;
    const int* ptr = p.get_adj_mat_ptr();
    int size = p.get_size();
    for (int i = 0; i < size; ++i)
        for (int j = i + 1; j < size; ++j)
            if (ptr[i * size + j] != 0)
                edge_num += 1;
    return edge_num;
}

constexpr int MAX_DEPTH = 5; // 非递归pattern matching支持的最大深度

template <int depth>
__device__ bool GPU_pattern_matching_func(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    uint32_t *edge, uint32_t* labeled_vertex, const char* p_label, GPUBitVector* fsm_set, int l_cnt)
{
    const int wid = threadIdx.x / THREADS_PER_WARP;
    const int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id   
    const int lid = threadIdx.x % THREADS_PER_WARP;
    int loop_set_prefix_id = schedule->get_loop_set_prefix_id(depth);
    int loop_size = vertex_set[loop_set_prefix_id].get_size();
    if (loop_size <= 0) //这个判断可能可以删了
        return false;
    uint32_t* loop_data_ptr = vertex_set[loop_set_prefix_id].get_data_ptr();

    bool local_match = false;
    __shared__ bool block_match[WARPS_PER_BLOCK];


    if (depth == schedule->get_size() - 1) {
        // warp 的线程一起做 insert
        for (int vertex_block = 0; vertex_block < loop_size; vertex_block += THREADS_PER_WARP)
        {
            if(vertex_block + lid >= loop_size) break;
            int vertex = loop_data_ptr[vertex_block + lid];
            if (subtraction_set.has_data_size(vertex, depth))
                continue;
            local_match = true;
            // for(int i = 0; i < WARPS_PER_BLOCK; i++) if(wid == i) {
                fsm_set[depth].insert(vertex);
            // }
            __threadfence_block();
        }
        __syncwarp();
        block_match[wid] = __any_sync(__activemask(), local_match);
        return block_match[wid]; 
    }

    for (int i = 0; i < loop_size; ++i)
    {
        uint32_t v = loop_data_ptr[i];
        if (subtraction_set.has_data_size(v, depth))
            continue;
        bool is_zero = false;
        for (int prefix_id = schedule->get_last(depth); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            unsigned int l, r;
            int target = schedule->get_prefix_target(prefix_id);
            get_labeled_edge_index(v, p_label[target], l, r);
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        if (lid == 0)
            subtraction_set.put(v, depth);
        __threadfence_block();

        if (GPU_pattern_matching_func<depth + 1>(schedule, vertex_set, subtraction_set, edge, labeled_vertex, p_label, fsm_set, l_cnt)) {
            local_match = true;
            if (threadIdx.x % THREADS_PER_WARP == 0) {
                fsm_set[depth].insert(v);
                __threadfence_block();
            }
        }
        __threadfence_block();
    }
    return local_match;
}

    template <>
__device__ bool GPU_pattern_matching_func<MAX_DEPTH>(const GPUSchedule* schedule, GPUVertexSet* vertex_set, GPUVertexSet& subtraction_set,
    uint32_t *edge, uint32_t* labeled_vertex, const char* p_label, GPUBitVector* fsm_set, int l_cnt)
{
    assert(false);
}


__global__ void gpu_single_pattern_matching(uint32_t job_id, uint32_t v_cnt, uint32_t buffer_size, uint32_t *edge, uint32_t* labeled_vertex, int* v_label, uint32_t* tmp, const GPUSchedule* schedule, char* all_p_label, GPUBitVector* global_fsm_set, unsigned int* label_start_idx, long long min_support, int l_cnt, bool* break_indicater){
    extern __shared__ GPUVertexSet block_vertex_set[];

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id within the block
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id   
    char* p_label = ((char*) (block_vertex_set)) + schedule->p_label_offset + (schedule->max_edge + 1) * wid; 
    int num_prefixes = schedule->get_total_prefix_num();
    int num_vertex_sets_per_warp = num_prefixes + 2;
    GPUVertexSet *vertex_set = block_vertex_set + wid * num_vertex_sets_per_warp;
    GPUBitVector *fsm_set = global_fsm_set + global_wid * schedule->get_size();
    // 这种 label 下开始的 vertex 和结束的 vertex 编号

    // FSM 使用的 vector，记录 support
    GPUVertexSet& subtraction_set = vertex_set[num_prefixes];
    
    if (lid == 0) {
        // set vertex_set's memory
        uint32_t offset = buffer_size * global_wid * num_vertex_sets_per_warp;
        for (int i = 0; i < num_vertex_sets_per_warp; ++i)
        {
            vertex_set[i].set_data_ptr(tmp + offset); // 注意这是个指针+整数运算，自带*4
            offset += buffer_size;
        }
        subtraction_set.init();
        size_t job_start_idx = job_id * schedule->get_size();
        for (int j = 0; j < schedule->get_size(); ++j)
            p_label[j] = all_p_label[job_start_idx + j];
    }

    if (lid < schedule->get_size())
        fsm_set[lid].clear();

    __threadfence_block();

    int start_v = label_start_idx[p_label[0]], end_v = label_start_idx[p_label[0] + 1];
    for(int vertex_block = start_v; vertex_block < end_v; vertex_block += num_total_warps) {
        // if(global_wid == 0 && lid == 0)
        //     printf("vertex_block: %d\n", vertex_block);
        int vertex_id = vertex_block + global_wid;
        if (vertex_id >= end_v) break;

        if(*break_indicater) break;

        bool is_zero = false;
        for (int prefix_id = schedule->get_last(0); prefix_id != -1; prefix_id = schedule->get_next(prefix_id)) {
            unsigned int l, r;
            int target = schedule->get_prefix_target(prefix_id);
            get_labeled_edge_index(vertex_id, p_label[target], l, r);
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], (int)r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        if (lid == 0)
            subtraction_set.put(vertex_id, 0);
        
        __threadfence_block();


        if(GPU_pattern_matching_func<1>(schedule, vertex_set, subtraction_set, edge, labeled_vertex, p_label, fsm_set, l_cnt)) {
            if(lid == 0){
                // for(int i = 0; i < num_total_warps; i++) if(i == global_wid) {
                    fsm_set[0].insert(vertex_id);
                // }
            }
        }

        __threadfence_block();

        // // try to cut down unneceseary ones

        // long long support = v_cnt;
        // for (int i = 0; i < schedule->size; i++){
        //     long long count = fsm_set[i].get_non_zero_cnt();
        //     if(count < support) support = count;
        // }
        // if(lid == 0) {
        //     if(support > min_support) {
        //         *break_indicater = true;
        //     }
        // }
    }

    // hipFree(temp_sum_counter);
}

class OrOperator{
    public:
    __device__ uint32_t operator() (const uint32_t &a, const uint32_t &b) const {
        return a | b;
    }
};

__global__ void reduce_fsm_set(GPUBitVector *global_fsm_set, uint32_t bit_vector_size, uint32_t schedule_size, int32_t *output) {
    int wid = threadIdx.x / THREADS_PER_WARP; // warp id within the block
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id  
    
    typedef hipcub::WarpReduce<uint32_t, THREADS_PER_WARP> WarpReduce;
    __shared__ typename WarpReduce::TempStorage temp_storage[WARPS_PER_BLOCK];
    // __shared__ uint32_t answer[THREADS_PER_BLOCK];

    // brute force version
    // for(int pos = 0; pos < schedule_size * bit_vector_size; pos++){
    //     int t = pos / bit_vector_size;
    //     int p = pos % bit_vector_size;    
    //     uint32_t tmp_result = 0;
    //     for(int i = 0; i < num_total_warps; i ++) {
    //         tmp_result |= (*(global_fsm_set + i * schedule_size + t)).get_data()[p];
    //         __syncwarp();
    //     }
    //     __threadfence();
    //     output[t].get_data()[p] = tmp_result;        
    // }

    for(int s_block = 0; s_block < schedule_size * bit_vector_size; s_block += num_total_warps) {
        int pos = s_block + global_wid;
        int t = pos / bit_vector_size;
        int p = pos % bit_vector_size;
        if(pos >= schedule_size * bit_vector_size) break;
        
        uint32_t tmp_result = 0;

        for(int i_block = 0; i_block < num_total_warps; i_block += WARPS_PER_BLOCK) {
            int i = i_block + lid;
            if(i >= num_total_warps) break;
            tmp_result |= (*(global_fsm_set + i * schedule_size + t)).get_data()[p];
        }
        __syncwarp();
        uint32_t agg = WarpReduce(temp_storage[wid]).Reduce(tmp_result, OrOperator());
        __syncwarp();
        if(lid == 0) {
            atomicAdd(&output[t], __popc(agg));
            __threadfence_system();
        }
    }
}



long long pattern_matching_init(const LabeledGraph *g, const Schedule_IEP& schedule, const std::vector<std::vector<int> >& automorphisms, unsigned int pattern_is_frequent_index, unsigned int* is_frequent, uint32_t* dev_edge, uint32_t* dev_labeled_vertex, int* dev_v_label, uint32_t* dev_tmp, int max_edge, int job_num, char* all_p_label, char* dev_all_p_label, GPUBitVector* dev_fsm_set, uint32_t* dev_label_start_idx, long long min_support) {

    printf("total prefix %d\n", schedule.get_total_prefix_num());
    schedule.print_schedule();
    fflush(stdout);

    tmpTime.check(); 

    long long sum = 0; //sum是这个pattern的所有labeled pattern中频繁的个数

    // create gpu schedule
    GPUSchedule* dev_schedule;
    gpuErrchk( hipMallocManaged((void**)&dev_schedule, sizeof(GPUSchedule)));
    dev_schedule->create_from_schedule(schedule);

    printf("schedule.prefix_num: %d\n", schedule.get_total_prefix_num());
    printf("shared memory for vertex set per block: %ld bytes\n", 
        (schedule.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet));


    tmpTime.print("Prepare time cost");
    tmpTime.check();

    uint32_t buffer_size = VertexSet::max_intersection_size;
    uint32_t block_shmem_size = (schedule.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + (max_edge + 1) * WARPS_PER_BLOCK * sizeof(char); // max_edge + 1是指一个pattern最多这么多点，用于存储p_label

    dev_schedule->p_label_offset = block_shmem_size - (max_edge + 1) * WARPS_PER_BLOCK * sizeof(char);
    dev_schedule->max_edge = max_edge;
    // 注意：此处没有错误，buffer_size代指每个顶点集所需的int数目，无需再乘sizeof(uint32_t)，但是否考虑对齐？
    //因为目前用了managed开内存，所以第一次运行kernel会有一定额外开销，考虑运行两次，第一次作为warmup
    
    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm, gpu_single_pattern_matching, THREADS_PER_BLOCK, block_shmem_size);
    printf("max number of active warps per SM: %d\n", max_active_blocks_per_sm * WARPS_PER_BLOCK);
    fflush(stdout);

    printf("total_job_num: %d\n", job_num);

    bool *break_indicater, result = false;
    gpuErrchk(hipMalloc((void**)&break_indicater, sizeof(bool)));
    int *dev_fsm_set_size;
    gpuErrchk(hipMallocManaged((void**)&dev_fsm_set_size, sizeof(int) * schedule.get_size()));

    for(int job_id = 0; job_id < job_num; job_id++){
        // if(job_id % 100 == 0)
        //     printf("job id: %d/%d\n",job_id, job_num);
        // fflush(stdout);

        gpuErrchk(hipMemcpy(break_indicater, &result, sizeof(bool), hipMemcpyHostToDevice));
        memset(dev_fsm_set_size, 0, sizeof(int) * schedule.get_size());

        // a *single* labeled pattern here
        gpu_single_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, block_shmem_size>>>(job_id, g->v_cnt, buffer_size, dev_edge, dev_labeled_vertex, dev_v_label, dev_tmp, dev_schedule, dev_all_p_label, dev_fsm_set, dev_label_start_idx, min_support, g->l_cnt, break_indicater);

        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk(hipMemcpy(&result, break_indicater, sizeof(bool), hipMemcpyDeviceToHost));

        long long support_answer = g->v_cnt;
        
        if(!result) {
            reduce_fsm_set<<<num_blocks, THREADS_PER_BLOCK>>>(dev_fsm_set, (g->v_cnt+31)/32, schedule.get_size(), dev_fsm_set_size);

            gpuErrchk( hipPeekAtLastError() );
            gpuErrchk( hipDeviceSynchronize() );
            for(int i = 0; i < schedule.get_size(); i++) {
                // printf("fsm_set[%d]:%lld\n", i, dev_fsm_set_size[i]);
                if(dev_fsm_set_size[i] < support_answer) support_answer = dev_fsm_set_size[i];
            }
            if(job_id % 100 == 0) {
                printf("finish job %d/%d, support answer:%lld\n", job_id, job_num, support_answer);
                fflush(stdout);
            }
        }

        if (result || support_answer >= min_support) {
            sum += 1;
            char* p_label = all_p_label + job_id * schedule.get_size(); 
            for (int aut_id = 0; aut_id < automorphisms.size(); ++aut_id) {
                const std::vector<int> & aut = automorphisms[aut_id];
                unsigned int index = pattern_is_frequent_index;
                unsigned int pow = 1;
                for (int j = 0; j < schedule.get_size(); ++j) {
                    index += p_label[ aut[j] ] * pow;
                    pow *= (unsigned int) g->l_cnt;
                }
                is_frequent[index >> 5] |= (1 << (index % 32));
            }
        }
    }

    printf("job_id: %d/%d sum:%lld\n", job_num, job_num, sum);
    fflush(stdout);

    gpuErrchk(hipFree(dev_schedule->father_prefix_id));
    gpuErrchk(hipFree(dev_schedule->last));
    gpuErrchk(hipFree(dev_schedule->next));
    gpuErrchk(hipFree(dev_schedule->loop_set_prefix_id));
    gpuErrchk(hipFree(dev_schedule->prefix_target));
    gpuErrchk(hipFree(dev_schedule));

    return sum;
}

void fsm_init(const LabeledGraph* g, int max_edge, int min_support) {
    std::vector<Pattern> patterns;
    Schedule_IEP* schedules;
    int schedules_num;
    int* mapping_start_idx;
    int* mappings;
    unsigned int* pattern_is_frequent_index; //每个unlabeled pattern对应的所有labeled pattern在is_frequent中的起始位置
    unsigned int* is_frequent; //bit vector
    g->get_fsm_necessary_info(patterns, max_edge, schedules, schedules_num, mapping_start_idx, mappings, pattern_is_frequent_index, is_frequent);
    long long fsm_cnt = 0;

    //特殊处理一个点的pattern
    for (int i = 0; i < g->l_cnt; ++i)
        if (g->label_frequency[i] >= min_support) {
            ++fsm_cnt;
            is_frequent[i >> 5] |= (unsigned int) (1 << (i % 32));
        }
    if (max_edge != 0)
        fsm_cnt = 0;
    int mapping_start_idx_pos = 1;

    size_t max_labeled_patterns = 1;
    for (int i = 0; i < max_edge + 1; ++i) //边数最大max_edge，点数最大max_edge + 1
        max_labeled_patterns *= (size_t) g->l_cnt;
    printf("max_labeled_patterns:%d\n", max_labeled_patterns);
    char* all_p_label = new char[max_labeled_patterns * (max_edge + 1) * 100];
    char* tmp_p_label = new char[(max_edge + 1) * 100];

    // 无关schedule的一些gpu初始化
    size_t size_edge = g->e_cnt * sizeof(uint32_t);
    size_t size_labeled_vertex = (g->v_cnt * g->l_cnt + 1) * sizeof(uint32_t);
    size_t size_v_label = g->v_cnt * sizeof(int);
    int max_total_prefix_num = 0;
    for (int i = 0; i < schedules_num; ++i)
    {
        schedules[i].update_loop_invariant_for_fsm();
        if (schedules[i].get_total_prefix_num() > max_total_prefix_num)
            max_total_prefix_num = schedules[i].get_total_prefix_num();
    }
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * (max_total_prefix_num + 2); //prefix + subtraction + tmp
    size_t size_all_p_label = max_labeled_patterns * (max_edge + 1) * sizeof(char);
    size_t size_label_start_idx = (g->l_cnt + 1) * sizeof(uint32_t);

    uint32_t *dev_edge;
    uint32_t *dev_labeled_vertex;
    int32_t *dev_v_label;
    uint32_t *dev_tmp;
    char *dev_all_p_label;
    uint32_t *dev_label_start_idx;
    GPUBitVector* dev_fsm_set;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_labeled_vertex, size_labeled_vertex));
    gpuErrchk( hipMalloc((void**)&dev_v_label, size_v_label));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));
    gpuErrchk( hipMalloc((void**)&dev_all_p_label, size_all_p_label));
    gpuErrchk( hipMalloc((void**)&dev_label_start_idx, size_label_start_idx));

    gpuErrchk( hipMemcpy(dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_labeled_vertex, g->labeled_vertex, size_labeled_vertex, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_v_label, g->v_label, size_v_label, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_label_start_idx, g->label_start_idx, size_label_start_idx, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_fsm_set, sizeof(GPUBitVector) * (num_total_warps + 1) * (max_edge + 1)));
    for (int i = 0; i < (num_total_warps + 1) * (max_edge + 1); ++i)
        dev_fsm_set[i].construct(g->v_cnt);

    timeval start, end, total_time;
    gettimeofday(&start, NULL);

    printf("schedule num: %d\n", schedules_num);


    for (int i = 1; i < schedules_num; ++i) {
        std::vector<std::vector<int> > automorphisms;
        automorphisms.clear();
        schedules[i].GraphZero_get_automorphisms(automorphisms);
        size_t all_p_label_idx = 0;
        g->traverse_all_labeled_patterns(schedules, all_p_label, tmp_p_label, mapping_start_idx, mappings, pattern_is_frequent_index, is_frequent, i, 0, mapping_start_idx_pos, all_p_label_idx);
        printf("all_p_label_idx: %u\n", all_p_label_idx);
        gpuErrchk( hipMemcpy(dev_all_p_label, all_p_label, all_p_label_idx * sizeof(char), hipMemcpyHostToDevice));
        int job_num = all_p_label_idx / schedules[i].get_size();

        fflush(stdout);

        fsm_cnt += pattern_matching_init(g, schedules[i], automorphisms, pattern_is_frequent_index[i], is_frequent, dev_edge, dev_labeled_vertex, dev_v_label, dev_tmp, max_edge, job_num, all_p_label, dev_all_p_label, dev_fsm_set, dev_label_start_idx, min_support);
        mapping_start_idx_pos += schedules[i].get_size();

        printf("temp fsm_cnt: %lld\n", fsm_cnt);

        if (get_pattern_edge_num(patterns[i]) != max_edge) //为了使得边数小于max_edge的pattern不被统计。正确性依赖于pattern按照边数排序
            fsm_cnt = 0;

        printf("fsm_cnt: %ld\n",fsm_cnt);

        // 时间相关
        gettimeofday(&end, NULL);
        timersub(&end, &start, &total_time);
        printf("time = %ld.%06ld s.\n", total_time.tv_sec, total_time.tv_usec);
    }

    printf("Counting time cost: %ld.%06ld s.\n", total_time.tv_sec, total_time.tv_usec);

    gpuErrchk(hipFree(dev_edge));
    //gpuErrchk(hipFree(dev_edge_from));
    gpuErrchk(hipFree(dev_labeled_vertex));
    gpuErrchk(hipFree(dev_v_label));
    gpuErrchk(hipFree(dev_tmp));
    gpuErrchk(hipFree(dev_all_p_label));
    gpuErrchk(hipFree(dev_label_start_idx));
    for (int i = 0; i < (max_edge + 1); ++i)
        dev_fsm_set[i].destroy();
    gpuErrchk(hipFree(dev_fsm_set));


    printf("fsm cnt = %lld\n", fsm_cnt);

    free(schedules);
    delete[] mapping_start_idx;
    delete[] mappings;
    delete[] pattern_is_frequent_index;
    delete[] is_frequent;
    delete[] all_p_label;
    delete[] tmp_p_label;
}

int main(int argc,char *argv[]) {
    printf("file_name: %s\n", argv[0]);
    print_parameter();
    LabeledGraph *g;
    DataLoader D;

    // const std::string type = argv[1];
    const std::string path = argv[1];
    const int max_edge = atoi(argv[2]);
    const int min_support = atoi(argv[3]);

    DataType my_type;
    
    GetDataType(my_type, "Patents");

    if(my_type == DataType::Invalid) {
        printf("Dataset not found!\n");
        return 0;
    }

    g = new LabeledGraph();
    assert(D.load_labeled_data(g,my_type,path.c_str())==true);

    fsm_init(g, max_edge, min_support);

    return 0;
}
