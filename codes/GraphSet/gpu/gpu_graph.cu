#include "hip/hip_runtime.h"
#include <algorithm>
#include <cassert>
#include <cstdint>
#include <cstring>
#include <string>

#include "common.h"
#include "dataloader.h"
#include "graph.h"
#include "motif_generator.h"
#include "schedule_IEP.h"
#include "vertex_set.h"
#include "timeinterval.h"

#include "component/gpu_device_detect.cuh"
#include "src/gpu_pattern_matching.cuh"


TimeInterval allTime;
TimeInterval tmpTime;

void pattern_matching(Graph *g, const Schedule_IEP &schedule_iep) {
    tmpTime.check();
    PatternMatchingDeviceContext *context;
    gpuErrchk(hipMallocManaged((void **)&context, sizeof(PatternMatchingDeviceContext)));
    context->init(g, schedule_iep);

    uint32_t buffer_size = VertexSet::max_intersection_size;
    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm, gpu_pattern_matching, THREADS_PER_BLOCK, context->block_shmem_size);
    fprintf(stderr, "max number of active warps per SM: %d\n", max_active_blocks_per_sm * WARPS_PER_BLOCK);

    tmpTime.print("Prepare time cost");
    tmpTime.check();

    unsigned long long sum = 0;

    gpu_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, context->block_shmem_size>>>(g->e_cnt, buffer_size, context);

    gpuErrchk(hipPeekAtLastError());
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipMemcpy(&sum, context->dev_sum, sizeof(sum), hipMemcpyDeviceToHost));

    sum /= schedule_iep.get_in_exclusion_optimize_redundancy();

    printf("Pattern count: %llu\n", sum);
    tmpTime.print("Counting time cost");

    context->destroy();
    gpuErrchk(hipFree(context));
}

int main(int argc, char *argv[]) {
    get_device_information();
    Graph *g;
    DataLoader D;

    if (argc < 4) {
        fprintf(stderr, "Usage: %s graph_file pattern_size pattern_string <1/0 for enable iep or not>\n", argv[0]);
        return 1;
    }

    int enable_iep = 1;
    if(argc >= 5) {
        enable_iep = atoi(argv[4]);
        if(enable_iep != 0 && enable_iep != 1) {
            fprintf(stderr, "Usage: %s graph_file pattern_size pattern_string <1/0 for enable iep or not>\n", argv[0]);
            return 1;
        } else {
            fprintf(stderr, "Enable iep: %d\n", enable_iep);
        }
    }

    using std::chrono::system_clock;
    auto t1 = system_clock::now();

    bool ok = D.fast_load(g, argv[1]);
    if (!ok) {
        fprintf(stderr, "data load failure :-(\n");
        return 0;
    }

    auto t2 = system_clock::now();
    auto load_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    fprintf(stderr, "Load data success! time: %g seconds\n", load_time.count() / 1.0e6);

    allTime.check();

    int pattern_size = atoi(argv[2]);
    const char *pattern_str = argv[3];

    Pattern p(pattern_size, pattern_str);

    printf("pattern = ");
    p.print();

    fprintf(stderr, "Max intersection size %d\n", VertexSet::max_intersection_size);

    tmpTime.check();
    
    bool is_pattern_valid;
    Schedule_IEP schedule_iep(p, is_pattern_valid, 1, 1, enable_iep, g->v_cnt, g->e_cnt, g->tri_cnt);
    if (!is_pattern_valid) {
        fprintf(stderr, "pattern is invalid!\n");
        return 1;
    }

    tmpTime.print("Schedule time cost");

    pattern_matching(g, schedule_iep);

    allTime.print("Total time cost");
    return 0;
}
