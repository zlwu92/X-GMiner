#include "hip/hip_runtime.h"
/**
 * 这个版本里面没有细粒度计时。有计时的在gpu_graph_with_timer.cu里面。
 * 而且计时的方式与zms版本略有区别。
*/
#define THRUST_IGNORE_CUB_VERSION_CHECK


#include <cassert>
#include <cstring>
#include <cstdint>
#include <string>
#include <algorithm>
#include <unordered_set>

#include <hipcub/hipcub.hpp>

#include "graph.h"
#include "dataloader.h"
#include "vertex_set.h"
#include "common.h"
#include "schedule_IEP.h"
#include "timeinterval.h"

#include "component/utils.cuh"
#include "component/gpu_schedule.cuh"

constexpr int THREADS_PER_BLOCK = 256;
constexpr int THREADS_PER_WARP = 32;
constexpr int WARPS_PER_BLOCK = THREADS_PER_BLOCK / THREADS_PER_WARP;

//#define PRINT_ANS_TO_FILE //用于scripts/small_graph_check.py

TimeInterval allTime;
TimeInterval tmpTime;

// __device__ void intersection1(uint32_t *tmp, uint32_t *lbases, uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size);
__device__ void intersection2(uint32_t *tmp, const uint32_t *lbases, const uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size);
static __device__ uint32_t do_intersection(uint32_t*, const uint32_t*, const uint32_t*, uint32_t, uint32_t);
class GPUVertexSet;
__device__ int unordered_subtraction_size(const GPUVertexSet& set0, const GPUVertexSet& set1, int size_after_restrict);




class GPUVertexSet
{
public:
    __device__ GPUVertexSet()
    {
        size = 0;
        data = NULL;
    }
    __device__ int get_size() const { return size;}
    __device__ uint32_t get_data(int i) const { return data[i];}
    __device__ void push_back(uint32_t val) { data[size++] = val;}
    __device__ void pop_back() { --size;}
    __device__ uint32_t get_last() const {return data[size - 1];}
    __device__ void set_data_ptr(uint32_t* ptr) { data = ptr;}
    __device__ uint32_t* get_data_ptr() const { return data;}
    __device__ bool has_data (uint32_t val) const // 注意：这里不用二分，调用它的是较小的无序集合
    {
        for (int i = 0; i < size; ++i)
            if (data[i] == val)
                return true;
        return false;
    }
    __device__ void init() { size = 0; }
    __device__ void init(uint32_t input_size, uint32_t* input_data)
    {
        size = input_size;
        data = input_data; //之后如果把所有prefix放到shared memory，由于input data在global memory上（因为是原图的边集），所以改成memcpy
    }
    __device__ void copy_from(const GPUVertexSet& other)//考虑改为并行
    {
        // 这个版本可能会有bank conflict
        uint32_t input_size = other.get_size(), *input_data = other.get_data_ptr();
        size = input_size;
        int lid = threadIdx.x % THREADS_PER_WARP; // lane id
        int size_per_thread = (input_size + THREADS_PER_WARP - 1) / THREADS_PER_WARP;
        int start = size_per_thread * lid;
        int end = min(start + size_per_thread, input_size);
        for (int i = start; i < end; ++i)
            data[i] = input_data[i];
        __threadfence_block();
    }
    __device__ void build_vertex_set(const GPUSchedule* schedule, const GPUVertexSet* vertex_set, uint32_t* input_data, uint32_t input_size, int prefix_id)
    {
        int father_id = schedule->get_father_prefix_id(prefix_id);
        if (father_id == -1)
        {
            if (threadIdx.x % THREADS_PER_WARP == 0)
                init(input_size, input_data);
            __threadfence_block();
        }
        else
        {
            // bool only_need_size = schedule->only_need_size[prefix_id];
            // if(only_need_size) {
            //     if (threadIdx.x % THREADS_PER_WARP == 0)
            //         init(input_size, input_data);
            //     __threadfence_block();
            //     if(input_size > vertex_set[father_id].get_size())
            //         this->size -= unordered_subtraction_size(*this, vertex_set[father_id], -1);
            //     else
            //         this->size = vertex_set[father_id].get_size() - unordered_subtraction_size(vertex_set[father_id], *this, -1);
            // }
            // else {
                intersection2(this->data, vertex_set[father_id].get_data_ptr(), input_data, vertex_set[father_id].get_size(), input_size, &this->size);
            // }
        }
    }


    __device__ void intersection_with(const GPUVertexSet& other)
    {
        uint32_t ret = do_intersection(data, data, other.get_data_ptr(), size, other.get_size());
        if (threadIdx.x % THREADS_PER_WARP == 0)
            size = ret;
        __threadfence_block();
    }

private:
    uint32_t size;
    uint32_t* data;
};

__device__ unsigned long long dev_sum = 0;
__device__ unsigned int dev_cur_edge = 0;

/**
 * search-based intersection
 * 
 * returns the size of the intersection set
 * 
 * @note：a和b并不是地位相等的。如果要进行in-place操作，请把输入放在a而不是b。
 * @todo：shared memory缓存优化
 */
__device__ uint32_t do_intersection(uint32_t* out, const uint32_t* a, const uint32_t* b, uint32_t na, uint32_t nb)
{
    // __shared__ uint32_t block_out_offset[THREADS_PER_BLOCK];
    __shared__ uint32_t block_out_size[WARPS_PER_BLOCK];
    typedef hipcub::WarpScan<uint8_t> WarpScan;
    __shared__ typename WarpScan::TempStorage temp_storage[WARPS_PER_BLOCK];

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    // uint32_t *out_offset = block_out_offset + wid * THREADS_PER_WARP;
    uint32_t &out_size = block_out_size[wid];

    if (lid == 0)
        out_size = 0;

    for(int num_done = 0; num_done < na; num_done += THREADS_PER_WARP) {
        bool found = 0;
        uint32_t u = 0;
        if (num_done + lid < na) {
            u = a[num_done + lid]; // u: an element in set a
            int mid, l = 0, r = int(nb) - 1;
            while (l <= r) {
                mid = (l + r) >> 1;
                if (b[mid] < u) {
                    l = mid + 1;
                } else if (b[mid] > u) {
                    r = mid - 1;
                } else {
                    found = 1;
                    break;
                }
            }
        }

        uint8_t thread_data = found;

        WarpScan(temp_storage[wid]).InclusiveSum(thread_data, thread_data);
        
        if (found) {
            out[out_size + thread_data - 1] = u;
        }

        // __syncwarp();

        if (lid == THREADS_PER_WARP - 1)
            out_size += thread_data;
    }

    __threadfence_block();
    return out_size;
}




__device__ void intersection2(uint32_t *tmp, const uint32_t *lbases, const uint32_t *rbases, uint32_t ln, uint32_t rn, uint32_t* p_tmp_size)
{
    // make sure ln <= rn
    if (ln > rn) {
        swap(ln, rn);
        swap(lbases, rbases);
    }
    /**
     * @todo 考虑ln < rn <= 32时，每个线程在lbases里面找rbases的一个元素可能会更快
     */

    uint32_t intersection_size = do_intersection(tmp, lbases, rbases, ln, rn);

    if (threadIdx.x % THREADS_PER_WARP == 0)
        *p_tmp_size = intersection_size;
    __threadfence_block();
}



constexpr int MAX_DEPTH = 10; // 非递归pattern matching支持的最大深度

template<int depth>
__device__ void GPU_pattern_matching_func(const GPUSchedule* schedule, GPUVertexSet* vertex_set, unsigned long long& local_ans, uint32_t *edge, e_index_t *vertex)
{

    if (depth == schedule->get_size() - schedule->get_in_exclusion_optimize_num()) {
        assert(false);
        // GPU_pattern_matching_final_in_exclusion(schedule, vertex_set, subtraction_set, tmp_set, local_ans,  edge, vertex);
        return;    
    }

    int loop_set_prefix_id = schedule->get_loop_set_prefix_id(depth);
    int loop_size = vertex_set[loop_set_prefix_id].get_size();

    uint32_t* loop_data_ptr = vertex_set[loop_set_prefix_id].get_data_ptr();
    if (depth == schedule->get_size() - 1 && schedule->get_in_exclusion_optimize_num() == 0) {
        local_ans += loop_size;
        return;
    }

    for (int i = 0; i < loop_size; ++i)
    {
        uint32_t v = loop_data_ptr[i];
        long long l, r;
        get_edge_index(v, l, r);
        bool is_zero = false;
        for (int prefix_id = schedule->get_last(depth); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], (int)(r - l), prefix_id);
            if (vertex_set[prefix_id].get_size() == schedule->get_break_size(prefix_id)) {
                is_zero = true;
                break;
            }
        }
        if (is_zero)
            continue;
        GPU_pattern_matching_func<depth + 1>(schedule, vertex_set, local_ans, edge, vertex);
    }
}

template <>
__device__ void GPU_pattern_matching_func<MAX_DEPTH>(const GPUSchedule* schedule, GPUVertexSet* vertex_set, unsigned long long& local_ans, uint32_t *edge, e_index_t *vertex)
{
    assert(false);
}

/**
 * @note `buffer_size`实际上是每个节点的最大邻居数量，而非所用空间大小
 */
__global__ void gpu_pattern_matching(uint32_t edge_num, uint32_t buffer_size, uint32_t *edge_from, uint32_t *edge, e_index_t *vertex, uint32_t *tmp, const GPUSchedule* schedule) {
    __shared__ unsigned int block_edge_idx[WARPS_PER_BLOCK];
    //之后考虑把tmp buffer都放到shared里来（如果放得下）
    extern __shared__ GPUVertexSet block_vertex_set[];
    
    int num_prefixes = schedule->get_total_prefix_num();
    int num_vertex_sets_per_warp = num_prefixes + 2;

    int wid = threadIdx.x / THREADS_PER_WARP; // warp id within the block
    int lid = threadIdx.x % THREADS_PER_WARP; // lane id
    int global_wid = blockIdx.x * WARPS_PER_BLOCK + wid; // global warp id
    unsigned int &edge_idx = block_edge_idx[wid];
    GPUVertexSet *vertex_set = block_vertex_set + wid * num_vertex_sets_per_warp;

    if (lid == 0) {
        edge_idx = 0;
        uint32_t offset = buffer_size * global_wid * num_vertex_sets_per_warp;
        for (int i = 0; i < num_vertex_sets_per_warp; ++i)
        {
            vertex_set[i].set_data_ptr(tmp + offset); // 注意这是个指针+整数运算，自带*4
            offset += buffer_size;
        }
    }
    // GPUVertexSet& subtraction_set = vertex_set[num_prefixes];
    // GPUVertexSet& tmp_set = vertex_set[num_prefixes + 1];

    __threadfence_block(); //之后考虑把所有的syncthreads都改成syncwarp


    uint32_t v0, v1;
    long long l, r;

    unsigned long long sum = 0;

    while (true) {
        if (lid == 0) {
            //if(++edgeI >= edgeEnd) { //这个if语句应该是每次都会发生吧？（是的
                edge_idx = atomicAdd(&dev_cur_edge, 1);
                //edgeEnd = min(edge_num, edgeI + 1); //这里不需要原子读吗
                // unsigned int i = edge_idx;
                // if (i < edge_num)
                // {
                //     subtraction_set.init();
                //     subtraction_set.push_back(edge_from[i]);
                //     subtraction_set.push_back(edge[i]);
                // }
            //}
        }

        __threadfence_block();

        unsigned int i = edge_idx;
        if(i >= edge_num) break;
    
    // for edge in E
        v0 = edge_from[i];
        v1 = edge[i];

        bool is_zero = false;
        get_edge_index(v0, l, r);
        for (int prefix_id = schedule->get_last(0); prefix_id != -1; prefix_id = schedule->get_next(prefix_id)) {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            // if(lid == 0){
            //     printf("edge:%d id:%d , size: %d\n",i, prefix_id, vertex_set[prefix_id].get_size());
            // }
            // vertex_set[prefix_id].get_size();
        }

        //目前只考虑pattern size>2的情况
        // start v1, depth = 1
        if (schedule->get_restrict_last(1) != -1 && v0 <= v1)
            continue;

        // v_0 > v_1, 合理
        
        get_edge_index(v1, l, r);
        for (int prefix_id = schedule->get_last(1); prefix_id != -1; prefix_id = schedule->get_next(prefix_id))
        {
            vertex_set[prefix_id].build_vertex_set(schedule, vertex_set, &edge[l], r - l, prefix_id);
            if (vertex_set[prefix_id].get_size() == 0 && prefix_id < schedule->get_basic_prefix_num()) {
                is_zero = true;
                break;
            }
            // if(lid == 0){
            //     printf("edge:%d id:%d , size: %d\n",i, prefix_id, vertex_set[prefix_id].get_size());
            // }
        }
        if (is_zero)
            continue;
        
        unsigned long long local_sum = 0; // local sum (corresponding to an edge index)
        GPU_pattern_matching_func<2>(schedule, vertex_set, local_sum, edge, vertex);
        // GPU_pattern_matching_aggressive_func(schedule, vertex_set, subtraction_set, tmp_set, local_sum, 2, edge, vertex);
        sum += local_sum;
    }



    if (lid == 0) {
        atomicAdd(&dev_sum, sum);
    }
}

void pattern_matching_init(Graph *g, const Schedule_IEP& schedule_iep) {
    printf("basic prefix %d, total prefix %d\n", schedule_iep.get_basic_prefix_num(), schedule_iep.get_total_prefix_num());

    int num_blocks = 1024;
    int num_total_warps = num_blocks * WARPS_PER_BLOCK;

    size_t size_edge = g->e_cnt * sizeof(v_index_t);
    size_t size_vertex = (g->v_cnt + 1) * sizeof(e_index_t);
    size_t size_tmp = VertexSet::max_intersection_size * sizeof(uint32_t) * num_total_warps * (schedule_iep.get_total_prefix_num() + 2); //prefix + subtraction + tmp

    schedule_iep.print_schedule();
    uint32_t *edge_from = new uint32_t[g->e_cnt];
    for(uint32_t i = 0; i < g->v_cnt; ++i)
        for(e_index_t j = g->vertex[i]; j < g->vertex[i+1]; ++j)
            edge_from[j] = i;

    tmpTime.check(); 

    uint32_t *dev_edge;
    uint32_t *dev_edge_from;
    e_index_t *dev_vertex;
    uint32_t *dev_tmp;

    gpuErrchk( hipMalloc((void**)&dev_edge, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_edge_from, size_edge));
    gpuErrchk( hipMalloc((void**)&dev_vertex, size_vertex));
    gpuErrchk( hipMalloc((void**)&dev_tmp, size_tmp));

    gpuErrchk( hipMemcpy(dev_edge, g->edge, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_edge_from, edge_from, size_edge, hipMemcpyHostToDevice));
    gpuErrchk( hipMemcpy(dev_vertex, g->vertex, size_vertex, hipMemcpyHostToDevice));

    unsigned long long sum = 0;
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_sum), &sum, sizeof(sum)));
    unsigned int cur_edge = 0;
    gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(dev_cur_edge), &cur_edge, sizeof(cur_edge)));

    //memcpy schedule
    GPUSchedule* dev_schedule;
    gpuErrchk( hipMallocManaged((void**)&dev_schedule, sizeof(GPUSchedule)));
    //dev_schedule->transform_in_exclusion_optimize_group_val(schedule);
    int schedule_size = schedule_iep.get_size();
    int max_prefix_num = schedule_size * (schedule_size - 1) / 2;
    
    bool *only_need_size = new bool[max_prefix_num];
    for(int i = 0; i < max_prefix_num; ++i)
        only_need_size[i] = schedule_iep.get_prefix_only_need_size(i);

    int in_exclusion_optimize_vertex_id_size = schedule_iep.in_exclusion_optimize_vertex_id.size();
    int in_exclusion_optimize_array_size  = schedule_iep.in_exclusion_optimize_coef.size();

    assert(in_exclusion_optimize_array_size == schedule_iep.in_exclusion_optimize_coef.size());
    assert(in_exclusion_optimize_array_size == schedule_iep.in_exclusion_optimize_flag.size());

    printf("array size %d\n", in_exclusion_optimize_array_size);
    fflush(stdout);

    int* in_exclusion_optimize_vertex_id = new int[in_exclusion_optimize_vertex_id_size];
    bool* in_exclusion_optimize_vertex_flag = new bool[in_exclusion_optimize_vertex_id_size];
    int* in_exclusion_optimize_vertex_coef = new int[in_exclusion_optimize_vertex_id_size];
    
    int* in_exclusion_optimize_coef = new int[in_exclusion_optimize_array_size];
    bool* in_exclusion_optimize_flag = new bool[in_exclusion_optimize_array_size];
    int* in_exclusion_optimize_ans_pos = new int[in_exclusion_optimize_array_size];

    for(int i = 0; i < in_exclusion_optimize_vertex_id_size; ++i) {
        in_exclusion_optimize_vertex_id[i] = schedule_iep.in_exclusion_optimize_vertex_id[i];
        in_exclusion_optimize_vertex_flag[i] = schedule_iep.in_exclusion_optimize_vertex_flag[i];
        in_exclusion_optimize_vertex_coef[i] = schedule_iep.in_exclusion_optimize_vertex_coef[i];
    }

    for(int i = 0; i < in_exclusion_optimize_array_size; ++i) {
        in_exclusion_optimize_coef[i] = schedule_iep.in_exclusion_optimize_coef[i];
        in_exclusion_optimize_flag[i] = schedule_iep.in_exclusion_optimize_flag[i];
        in_exclusion_optimize_ans_pos[i] = schedule_iep.in_exclusion_optimize_ans_pos[i];
    }

    if (in_exclusion_optimize_vertex_id_size > 0) {
        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_id, in_exclusion_optimize_vertex_id, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));
        
        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_flag, in_exclusion_optimize_vertex_flag, sizeof(bool) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));
        
        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_vertex_coef, in_exclusion_optimize_vertex_coef, sizeof(int) * in_exclusion_optimize_vertex_id_size, hipMemcpyHostToDevice));
    }

    if (in_exclusion_optimize_array_size > 0)
    {
        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_coef, in_exclusion_optimize_coef, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));

        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_flag, in_exclusion_optimize_flag, sizeof(bool) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));
        
        gpuErrchk( hipMallocManaged((void**)&dev_schedule->in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size));
        gpuErrchk( hipMemcpy(dev_schedule->in_exclusion_optimize_ans_pos, in_exclusion_optimize_ans_pos, sizeof(int) * in_exclusion_optimize_array_size, hipMemcpyHostToDevice));
    }

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->adj_mat, sizeof(int) * schedule_size * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->adj_mat, schedule_iep.get_adj_mat_ptr(), sizeof(int) * schedule_size * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->father_prefix_id, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->father_prefix_id, schedule_iep.get_father_prefix_id_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->last, schedule_iep.get_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->next, schedule_iep.get_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->only_need_size, sizeof(bool) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->only_need_size, only_need_size, sizeof(bool) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->break_size, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->break_size, schedule_iep.get_break_size_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->loop_set_prefix_id, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->loop_set_prefix_id, schedule_iep.get_loop_set_prefix_id_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));

    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_last, sizeof(int) * schedule_size));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_last, schedule_iep.get_restrict_last_ptr(), sizeof(int) * schedule_size, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_next, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_next, schedule_iep.get_restrict_next_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));
    
    gpuErrchk( hipMallocManaged((void**)&dev_schedule->restrict_index, sizeof(int) * max_prefix_num));
    gpuErrchk( hipMemcpy(dev_schedule->restrict_index, schedule_iep.get_restrict_index_ptr(), sizeof(int) * max_prefix_num, hipMemcpyHostToDevice));

    dev_schedule->in_exclusion_optimize_array_size = in_exclusion_optimize_array_size;
    dev_schedule->in_exclusion_optimize_vertex_id_size = in_exclusion_optimize_vertex_id_size;
    dev_schedule->size = schedule_iep.get_size();
    dev_schedule->total_prefix_num = schedule_iep.get_total_prefix_num();
    dev_schedule->basic_prefix_num = schedule_iep.get_basic_prefix_num();
    dev_schedule->total_restrict_num = schedule_iep.get_total_restrict_num();
    dev_schedule->in_exclusion_optimize_num = schedule_iep.get_in_exclusion_optimize_num();
    //dev_schedule->k_val = schedule.get_k_val();

    printf("schedule_iep.prefix_num: %d\n", schedule_iep.get_total_prefix_num());
    printf("shared memory for vertex set per block: %ld bytes\n", 
        (schedule_iep.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int));

    tmpTime.print("Prepare time cost");
    tmpTime.check();

    uint32_t buffer_size = VertexSet::max_intersection_size;
    uint32_t block_shmem_size = (schedule_iep.get_total_prefix_num() + 2) * WARPS_PER_BLOCK * sizeof(GPUVertexSet) + in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);
    dev_schedule->ans_array_offset = block_shmem_size - in_exclusion_optimize_vertex_id_size * WARPS_PER_BLOCK * sizeof(int);
    // 注意：此处没有错误，buffer_size代指每个顶点集所需的int数目，无需再乘sizeof(uint32_t)，但是否考虑对齐？
    //因为目前用了managed开内存，所以第一次运行kernel会有一定额外开销，考虑运行两次，第一次作为warmup
    
    int max_active_blocks_per_sm;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_active_blocks_per_sm, gpu_pattern_matching, THREADS_PER_BLOCK, block_shmem_size);
    printf("max number of active warps per SM: %d\n", max_active_blocks_per_sm * WARPS_PER_BLOCK);
    
    gpu_pattern_matching<<<num_blocks, THREADS_PER_BLOCK, block_shmem_size>>>
        (g->e_cnt, buffer_size, dev_edge_from, dev_edge, dev_vertex, dev_tmp, dev_schedule);

    gpuErrchk( hipPeekAtLastError() );
    gpuErrchk( hipDeviceSynchronize() );


    gpuErrchk( hipMemcpyFromSymbol(&sum, HIP_SYMBOL(dev_sum), sizeof(sum)) );

    sum /= schedule_iep.get_in_exclusion_optimize_redundancy();

    printf("%d\n",schedule_iep.get_in_exclusion_optimize_redundancy());
    
    #ifdef PRINT_ANS_TO_FILE
    freopen("1.out", "w", stdout);
    printf("count %llu\n", sum);
    fclose(stdout);
    #endif
    printf("count %llu\n", sum);
    tmpTime.print("Counting time cost");
    //之后需要加上hipFree

    // 尝试释放一些内存
    gpuErrchk(hipFree(dev_edge));
    gpuErrchk(hipFree(dev_edge_from));
    gpuErrchk(hipFree(dev_vertex));
    gpuErrchk(hipFree(dev_tmp));

    gpuErrchk(hipFree(dev_schedule->adj_mat));
    gpuErrchk(hipFree(dev_schedule->father_prefix_id));
    gpuErrchk(hipFree(dev_schedule->last));
    gpuErrchk(hipFree(dev_schedule->next));
    gpuErrchk(hipFree(dev_schedule->loop_set_prefix_id));
    gpuErrchk(hipFree(dev_schedule->restrict_last));
    gpuErrchk(hipFree(dev_schedule->restrict_next));
    gpuErrchk(hipFree(dev_schedule->restrict_index));

    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_vertex_id));
    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_coef));
    gpuErrchk(hipFree(dev_schedule->in_exclusion_optimize_flag));

    gpuErrchk(hipFree(dev_schedule));

    delete[] edge_from;
    delete[] in_exclusion_optimize_vertex_id;
    delete[] in_exclusion_optimize_coef;
    delete[] in_exclusion_optimize_flag;
    delete[] only_need_size;
}


int main(int argc,char *argv[]) {
    Graph *g;
    DataLoader D;

    using std::chrono::system_clock;
    auto t1 = system_clock::now();
    
    if(argc != 3) {
        printf("usage: %s graph_file clique_size", argv[0]);
        return 0;
    }

    // 读入 pattern size
    int pattern_size = atoi(argv[2]);

    Pattern p(pattern_size);

    for(int i = 0; i < pattern_size; i++) {
        for(int j = i + 1; j < pattern_size; j++) {
            p.add_edge(i, j);
        }
    }

    DataType type = DataType::Patents;

    bool ok = D.fast_load(g, argv[1]);

    // bool ok = D.load_data(g, type, argv[1]);

    if (!ok) {
        printf("data load failure :-(\n");
        return 0;
    }


    reduce_edges_for_clique(*g);
    // warm up GPU
    hipFree(0);


    auto t2 = system_clock::now();
    auto load_time = std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1);
    printf("Load data success! time: %g seconds\n", load_time.count() / 1.0e6);
    fflush(stdout);

    allTime.check();


    printf("pattern = \n");
    p.print();
    printf("max intersection size %d\n", VertexSet::max_intersection_size);
    bool is_pattern_valid = true;
    bool use_in_exclusion_optimize = true;
    Schedule_IEP schedule_iep(p.get_adj_mat_ptr(), pattern_size);
    // Schedule_IEP schedule_iep(p, is_pattern_valid, 1, 1, use_in_exclusion_optimize, g->v_cnt, g->e_cnt, g->tri_cnt);
    schedule_iep.set_in_exclusion_optimize_redundancy(1);

    if (!is_pattern_valid) {
        printf("pattern is invalid!\n");
        return 0;
    }

    pattern_matching_init(g, schedule_iep);

    allTime.print("Total time cost");

    return 0;
}