#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "pattern.hh"
#include "operations.cuh"
#include "hip/hip_runtime_api.h"
#include "cuda_launch_config.hpp"

#define BLK_SZ BLOCK_SIZE
typedef hipcub::BlockReduce<AccType, BLK_SZ> BlockReduce;
#include "warp_edge_iterative.cuh"

void SglSolver(Graph &g, Pattern &p, uint64_t &total, int, int) {
  size_t memsize = print_device_info(0);
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  GraphGPU gg(g);
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  int k = 4;
  if (p.is_house() || p.is_pentagon()) k = 5;

  size_t nthreads = BLK_SZ;
  size_t nwarps = BLK_SZ/WARP_SIZE;
  size_t ntasks = nv;
  ntasks = gg.init_edgelist(g, 1);
  std::cout << "Edge parallel: edgelist size = " << ntasks << "\n";
  size_t nblocks = (ntasks-1)/nwarps+1;
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_edge_iterative, nthreads, 0);
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  if (p.is_diamond() || p.is_pentagon()) nblocks = std::min(3*max_blocks, nblocks);
  std::cout << p.get_name() << " max_blocks_per_SM = " << max_blocks_per_SM << "\n";

  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
  size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
  auto nb = int64_t(memsize - mem_graph) / int64_t(per_block_vlist_size);
  if (nb < nblocks) nblocks = nb;
  size_t list_size = nblocks * per_block_vlist_size;
  if (p.is_rectangle() || p.is_pentagon()) list_size = 0;
  std::cout << "frontier list size " << list_size/(1024*1024) << " MB\n";
  if (list_size > 0) CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));
  std::cout << "CUDA subgraph listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA) ...\n";

  Timer t;
  t.Start();
  hipProfilerStart();
  warp_edge_iterative<<<nblocks, nthreads>>>(ntasks, gg, orders, frontier_list, md, d_total);
  hipProfilerStop();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [cuda_base] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

