#include "hip/hip_runtime.h"
// Copyright 2020 MIT
// Authors: Xuhao Chen <cxh@mit.edu>

#include "fsm.h"
#include "timer.h"
#include "cutil_subset.h"
#include <hipcub/hipcub.hpp>
#define USE_PID
#define USE_DOMAIN
#define EDGE_INDUCED
#define ENABLE_LABEL
#include "pangolin_gpu/miner.cuh"
#include "bitsets.h"
#include "fsm_operations.cuh"
#include <thrust/scan.h>
#include <thrust/extrema.h>
#include <thrust/execution_policy.h>

__global__ void extend_alloc(unsigned m, unsigned level, GraphGPU graph, EmbeddingList emb_list, IndexT *num_new_emb) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vid[BLOCK_SIZE][MAX_SIZE];
  __shared__ BYTE his[BLOCK_SIZE][MAX_SIZE];
  if(pos < m) {
    emb_list.get_edge_embedding(level, pos, vid[tid], his[tid]);
    num_new_emb[pos] = 0;
    for (unsigned i = 0; i < level+1; ++i) {
      IndexT src = vid[tid][i];
      IndexT row_begin = graph.edge_begin(src);
      IndexT row_end = graph.edge_end(src);
      for (IndexT e = row_begin; e < row_end; e++) {
        IndexT dst = graph.getEdgeDst(e);
        if (!is_edge_automorphism(level+1, vid[tid], his[tid], i, src, dst))
          num_new_emb[pos] ++;
      }
    }
  }
}

__global__ void extend_insert(unsigned m, unsigned level, GraphGPU graph, EmbeddingList emb_list, IndexT *indices) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][MAX_SIZE];
  __shared__ BYTE his[BLOCK_SIZE][MAX_SIZE];
  if(pos < m) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    IndexT start = indices[pos];
    for (unsigned i = 0; i < level+1; ++i) {
      IndexT src = vids[tid][i];
      IndexT row_begin = graph.edge_begin(src);
      IndexT row_end = graph.edge_end(src);
      for (IndexT e = row_begin; e < row_end; e++) {
        IndexT dst = graph.getEdgeDst(e);
        if (!is_edge_automorphism(level+1, vids[tid], his[tid], i, src, dst)) {
          emb_list.set_idx(level+1, start, pos);
          emb_list.set_his(level+1, start, i);
          emb_list.set_vid(level+1, start++, dst);
        }
      }
    }
  }
}

__global__ void init_aggregate(unsigned m, unsigned num_emb, GraphGPU graph, EmbeddingList emb_list, unsigned *pids, int nlabels, unsigned threshold, Bitsets small_sets, Bitsets large_sets) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < num_emb) {
    IndexT src = emb_list.get_idx(1, pos);
    IndexT dst = emb_list.get_vid(1, pos);
    BYTE src_label = graph.getData(src);
    BYTE dst_label = graph.getData(dst);
    int pid = 0;
    if (src_label <= dst_label)
      pid = get_init_pattern_id(src_label, dst_label, nlabels);
    else pid = get_init_pattern_id(dst_label, src_label, nlabels);
    pids[pos] = pid;
    if (src_label < dst_label) {
      small_sets.set(pid, src);
      large_sets.set(pid, dst);
    } else if (src_label > dst_label) {
      small_sets.set(pid, dst);
      large_sets.set(pid, src);
    } else {
      small_sets.set(pid, src);
      small_sets.set(pid, dst);
      large_sets.set(pid, src);
      large_sets.set(pid, dst);
    }
  }
}

__global__ void count_ones(int id, Bitsets sets, int *count) {
  typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int num = 0;
  if(pos < sets.vec_size())
    num = sets.count_num_ones(id, pos);
  int block_total = BlockReduce(temp_storage).Sum(num);
  if(threadIdx.x == 0) atomicAdd(count, block_total);
}

int init_support_count(unsigned m, int npatterns, unsigned threshold, Bitsets small_sets, Bitsets large_sets, bool *init_support_map) {
  int num_freq_patterns = 0;
  for (int i = 0; i < npatterns; i++) {
    int a, b, *d_count;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_count, sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m-1)/256+1, 256>>>(i, small_sets, d_count);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(&a, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m-1)/256+1, 256>>>(i, large_sets, d_count);
    CUDA_SAFE_CALL(hipMemcpy(&b, d_count, sizeof(int), hipMemcpyDeviceToHost));
    unsigned support = a < b ? a : b;
    if (support >= threshold) {
      init_support_map[i] = 1;
      num_freq_patterns ++;
    } else init_support_map[i] = 0;
  }
  return num_freq_patterns;
}

//int support_count(unsigned m, unsigned npatterns, unsigned threshold, SetType *small_sets, SetType *middle_sets, SetType *large_sets, bool *support_map) {
int support_count(unsigned m, unsigned npatterns, unsigned threshold, Bitsets small_sets, Bitsets middle_sets, Bitsets large_sets, bool *support_map) {
  int num_freq_patterns = 0;
  for (int i = 0; i < npatterns; i++) {
    int a, b, c, *d_count;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_count, sizeof(int)));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m-1)/256+1, 256>>>(i, small_sets, d_count);
    CUDA_SAFE_CALL(hipMemcpy(&a, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m-1)/256+1, 256>>>(i, large_sets, d_count);
    CUDA_SAFE_CALL(hipMemcpy(&b, d_count, sizeof(int), hipMemcpyDeviceToHost));
    CUDA_SAFE_CALL(hipMemset(d_count, 0, sizeof(int)));
    count_ones<<<(m-1)/256+1, 256>>>(i, middle_sets, d_count);
    CUDA_SAFE_CALL(hipMemcpy(&c, d_count, sizeof(int), hipMemcpyDeviceToHost));
    unsigned small = a < b ? a : b;
    unsigned support = small < c ? small : c;
    if (support >= threshold) {
      support_map[i] = 1;
      num_freq_patterns ++;
    } else support_map[i] = 0;
  }
  return num_freq_patterns;
}

__global__ void init_filter_check(unsigned m, unsigned *pids, bool *init_support_map, IndexT *is_frequent_emb) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < m) {
    unsigned pid = pids[pos];
    bool is_frequent = init_support_map[pid];
    if (is_frequent) is_frequent_emb[pos] = 1;
  }
}

__global__ void copy_vids(unsigned m, EmbeddingList emb_list, IndexT *vid_list0, IndexT *vid_list1) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < m) {
    vid_list0[pos] = emb_list.get_idx(1, pos);
    vid_list1[pos] = emb_list.get_vid(1, pos);
  }
}

__global__ void init_filter(unsigned m, EmbeddingList emb_list, IndexT *vid_list0, IndexT *vid_list1, IndexT *indices, IndexT *is_frequent_emb) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if(pos < m) {
    if (is_frequent_emb[pos]) {
      IndexT src = vid_list0[pos];
      IndexT dst = vid_list1[pos];
      unsigned start = indices[pos];
      emb_list.set_vid(1, start, dst);
      emb_list.set_idx(1, start, src);
    }
  }
}

__global__ void aggregate_check(unsigned num_emb, unsigned level, GraphGPU graph, EmbeddingList emb_list, unsigned *pids, int nlabels, unsigned threshold, unsigned *ne) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][MAX_SIZE];
  __shared__ BYTE his[BLOCK_SIZE][MAX_SIZE];
  if(pos < num_emb) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    unsigned n = level+1;
    assert(n < 4);
    IndexT first = vids[tid][0];
    IndexT second = vids[tid][1];
    IndexT third = vids[tid][2];
    BYTE l0 = graph.getData(first);
    BYTE l1 = graph.getData(second);
    BYTE l2 = graph.getData(third);
    BYTE h2 = his[tid][2];
    unsigned pid = 0;
    if (n == 3) {
      if (h2 == 0) {
        if (l1 < l2) {
          pid = get_pattern_id(l0, l2, l1, nlabels);
        } else {
          pid = get_pattern_id(l0, l1, l2, nlabels);
        }
      } else {
        assert(h2 == 1);
        if (l0 < l2) {
          pid = get_pattern_id(l1, l2, l0, nlabels);
        } else {
          pid = get_pattern_id(l1, l0, l2, nlabels);
        }
      }
    } else {
    }
    pids[pos] = pid;
    atomicAdd(&ne[pid], 1);
  }
}

__global__ void find_candidate_patterns(unsigned num_patterns, unsigned *ne, unsigned minsup, unsigned *id_map, unsigned *num_new_patterns) {
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  if (pos < num_patterns) {
    if (ne[pos] >= minsup) {
      unsigned new_id = atomicAdd(num_new_patterns, 1);
      id_map[pos] = new_id;
    }
  }
}

__global__ void aggregate(unsigned m, unsigned num_emb, unsigned level, GraphGPU graph, EmbeddingList emb_list, unsigned *pids, unsigned *ne, unsigned *id_map, int nlabels, unsigned threshold, Bitsets small_sets, Bitsets middle_sets, Bitsets large_sets) {
  unsigned tid = threadIdx.x;
  unsigned pos = blockIdx.x * blockDim.x + threadIdx.x;
  __shared__ IndexT vids[BLOCK_SIZE][MAX_SIZE];
  __shared__ BYTE his[BLOCK_SIZE][MAX_SIZE];
  if(pos < num_emb) {
    emb_list.get_edge_embedding(level, pos, vids[tid], his[tid]);
    unsigned n = level+1;
    assert(n == 3);
    IndexT first = vids[tid][0];
    IndexT second = vids[tid][1];
    IndexT third = vids[tid][2];
    BYTE l0 = graph.getData(first);
    BYTE l1 = graph.getData(second);
    BYTE l2 = graph.getData(third);
    BYTE h2 = his[tid][2];
    IndexT small, middle, large;
    unsigned pid = pids[pos];
    if (ne[pid] >= threshold) {
      pid = id_map[pid];
      if (h2 == 0) {
        middle = first;
        if (l1 < l2) {
          small = second;
          large = third;
        } else {
          small = third;
          large = second;
        }
        small_sets.set(pid, small);
        middle_sets.set(pid, middle);
        large_sets.set(pid, large);
        if (l1 == l2) {
          small_sets.set(pid, large);
          large_sets.set(pid, small);
        }
      } else {
        assert(h2 == 1);
        middle = second;
        if (l0 < l2) {
          small = first;
          large = third;
        } else {
          small = third;
          large = first;
        }
        small_sets.set(pid, small);
        middle_sets.set(pid, middle);
        large_sets.set(pid, large);
        if (l0 == l2) {
          small_sets.set(pid, large);
          large_sets.set(pid, small);
        }
      }
    }
  }
}

void parallel_prefix_sum(int n, IndexT *in, IndexT *out) {
  IndexT total = 0;
  for (size_t i = 0; i < n; i++) {
    out[i] = total;
    total += in[i];
  }
  out[n] = total;
}

void FsmSolver(Graph &g, unsigned k, unsigned minsup, int nlabels, int &total_num) {
  print_device_info(0);
  int m = g.num_vertices();
  int nnz = g.num_edges();
  int nthreads = BLOCK_SIZE;
  int nblocks = DIVIDE_INTO(nnz, nthreads);

  GraphGPU gg(g);
  EmbeddingList emb_list;
  emb_list.init(nnz, k+1, false);
  emb_list.init_cpu(g);

  int num_init_patterns = (nlabels+1)*(nlabels+1);
  std::cout << "Number of init patterns: " << num_init_patterns << std::endl;
  unsigned num_emb = emb_list.size();
  std::cout << "number of single-edge embeddings: " << num_emb << "\n";
  unsigned *pids;
  CUDA_SAFE_CALL(hipMalloc((void **)&pids, sizeof(unsigned)*num_emb));
  bool *h_init_support_map = (bool *)malloc(sizeof(bool) * num_init_patterns);
  bool *d_init_support_map;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_init_support_map, sizeof(bool)*num_init_patterns));
  IndexT *is_frequent_emb;
  CUDA_SAFE_CALL(hipMalloc((void **)&is_frequent_emb, sizeof(IndexT)*(num_emb+1)));
  CUDA_SAFE_CALL(hipMemset(is_frequent_emb, 0, sizeof(IndexT)*(num_emb+1)));
  IndexT *vid_list0, *vid_list1;
  CUDA_SAFE_CALL(hipMalloc((void **)&vid_list0, sizeof(IndexT)*num_emb));
  CUDA_SAFE_CALL(hipMalloc((void **)&vid_list1, sizeof(IndexT)*num_emb));
  Bitsets small_sets, large_sets, middle_sets;
  small_sets.alloc(MAX_NUM_PATTERNS, m);
  large_sets.alloc(MAX_NUM_PATTERNS, m);
  middle_sets.alloc(MAX_NUM_PATTERNS, m);
  small_sets.set_size(num_init_patterns, m);
  large_sets.set_size(num_init_patterns, m);
  middle_sets.set_size(num_init_patterns, m);

  IndexT *num_new_emb, *indices;
  CUDA_SAFE_CALL(hipMalloc((void **)&indices, sizeof(IndexT) * (num_emb+1)));
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  nblocks = (num_emb-1)/nthreads+1;
  unsigned *d_num_new_patterns;
  unsigned h_num_new_patterns = 0;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_num_new_patterns, sizeof(unsigned)));
  std::cout << "CUDA " << k << "-FSM (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  Timer t;
  t.Start();
  unsigned level = 1;
  init_aggregate<<<nblocks, nthreads>>>(m, num_emb, gg, emb_list, pids, nlabels, minsup, small_sets, large_sets);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  //std::cout << "Init_aggregate Done\n";
  int num_freq_patterns = init_support_count(m, num_init_patterns, minsup, small_sets, large_sets, h_init_support_map);
  total_num += num_freq_patterns;
  if (num_freq_patterns == 0) {
    std::cout << "No frequent pattern found\n\n";
    return;
  }
  std::cout << "Number of frequent single-edge patterns: " << num_freq_patterns << "\n";
  CUDA_SAFE_CALL(hipMemcpy(d_init_support_map, h_init_support_map, sizeof(bool) * num_init_patterns, hipMemcpyHostToDevice));
  init_filter_check<<<nblocks, nthreads>>>(num_emb, pids, d_init_support_map, is_frequent_emb);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  thrust::exclusive_scan(thrust::device, is_frequent_emb, is_frequent_emb+num_emb+1, indices);
  IndexT new_size;
  CUDA_SAFE_CALL(hipMemcpy(&new_size, &indices[num_emb], sizeof(IndexT), hipMemcpyDeviceToHost));
  std::cout << "number of embeddings after pruning: " << new_size << "\n";
  copy_vids<<<nblocks, nthreads>>>(num_emb, emb_list, vid_list0, vid_list1);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  init_filter<<<nblocks, nthreads>>>(num_emb, emb_list, vid_list0, vid_list1, indices, is_frequent_emb);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  CUDA_SAFE_CALL(hipFree(indices));
  CUDA_SAFE_CALL(hipFree(is_frequent_emb));
  CUDA_SAFE_CALL(hipFree(pids));
  //small_sets.clean();
  //large_sets.clean();
  small_sets.clear();
  large_sets.clear();
  CUDA_SAFE_CALL(hipFree(vid_list0));
  CUDA_SAFE_CALL(hipFree(vid_list1));
  CUDA_SAFE_CALL(hipFree(d_init_support_map));
  emb_list.remove_tail(new_size);

  while (1) {
    num_emb = emb_list.size();
    std::cout << "number of embeddings in level " << level << ": " << num_emb << "\n";
    CUDA_SAFE_CALL(hipMalloc((void **)&num_new_emb, sizeof(IndexT) * (num_emb+1)));
    CUDA_SAFE_CALL(hipMalloc((void **)&indices, sizeof(IndexT) * (num_emb+1)));
    //std::cout << "Done allocating memory for embeddings in level " << level << "\n";
    nblocks = (num_emb-1)/nthreads+1;
    extend_alloc<<<nblocks, nthreads>>>(num_emb, level, gg, emb_list, num_new_emb);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    //std::cout << "Extend_alloc Done\n";
    thrust::exclusive_scan(thrust::device, num_new_emb, num_new_emb+num_emb+1, indices);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    //std::cout << "PrefixSum Done\n";
    CUDA_SAFE_CALL(hipMemcpy(&new_size, &indices[num_emb], sizeof(IndexT), hipMemcpyDeviceToHost));
    assert(new_size < 4294967296); // TODO: currently do not support vector size larger than 2^32
    std::cout << "number of new embeddings: " << new_size << "\n";
    emb_list.add_level(new_size);
    extend_insert<<<nblocks, nthreads>>>(num_emb, level, gg, emb_list, indices);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    //std::cout << "Extend_insert Done\n";
    num_emb = emb_list.size();
    CUDA_SAFE_CALL(hipFree(num_new_emb));
    CUDA_SAFE_CALL(hipFree(indices));
    level ++;

    int num_patterns = nlabels*num_init_patterns;
    nblocks = (num_emb-1)/nthreads+1;
    std::cout << "Number of patterns in level " << level << ": " << num_patterns << std::endl;
    std::cout << "number of embeddings in level " << level << ": " << num_emb << "\n";
    unsigned *ne, *id_map;
    CUDA_SAFE_CALL(hipMalloc((void **)&ne, sizeof(unsigned)*num_patterns));
    CUDA_SAFE_CALL(hipMalloc((void **)&id_map, sizeof(unsigned)*num_patterns));
    CUDA_SAFE_CALL(hipMemset(ne, 0, sizeof(unsigned)*num_patterns));
    CUDA_SAFE_CALL(hipMalloc((void **)&pids, sizeof(unsigned)*num_emb));
    //std::cout << "Done allocating memory for aggregation in level " << level << "\n";
    aggregate_check<<<nblocks, nthreads>>>(num_emb, level, gg, emb_list, pids, nlabels, minsup, ne);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemset(d_num_new_patterns, 0, sizeof(unsigned)));
    find_candidate_patterns<<<(num_patterns-1)/nthreads+1, nthreads>>>(num_patterns, ne, minsup, id_map, d_num_new_patterns);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(&h_num_new_patterns, d_num_new_patterns, sizeof(unsigned), hipMemcpyDeviceToHost));
    std::cout << "Number of candidate patterns in level " << level << ": " << h_num_new_patterns << std::endl;

    //small_sets.alloc(h_num_new_patterns, m);
    //large_sets.alloc(h_num_new_patterns, m);
    //middle_sets.alloc(h_num_new_patterns, m);
    small_sets.set_size(h_num_new_patterns, m);
    large_sets.set_size(h_num_new_patterns, m);
    middle_sets.set_size(h_num_new_patterns, m);
    //std::cout << "Done allocating sets\n";
    aggregate<<<nblocks, nthreads>>>(m, num_emb, level, gg, emb_list, pids, ne, id_map, nlabels, minsup, small_sets, middle_sets, large_sets);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    bool *h_support_map = (bool *)malloc(sizeof(bool) * h_num_new_patterns);
    num_freq_patterns = support_count(m, h_num_new_patterns, minsup, small_sets, middle_sets, large_sets, h_support_map);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipFree(ne));
    CUDA_SAFE_CALL(hipFree(id_map));
    std::cout << "num_frequent_patterns: " << num_freq_patterns << "\n";
    total_num += num_freq_patterns;
    if (num_freq_patterns == 0) break;
    if (level == k) break;
    //filter<<<nblocks, nthreads>>>(level, emb_list);
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [gpu_base] = " << t.Seconds() << " sec\n";
}

