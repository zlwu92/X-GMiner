#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"
//#define FISSION
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

#include "motif3_edge_warp.cuh"
#include "wedge_edge_warp.cuh"
#include "triangle_edge_warp.cuh"
#include "motif4_edge_warp.cuh"
#include "motif4_edge_warp_fission.cuh"

__global__ void clear(AccType *accumulators) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  accumulators[i] = 0;
}

void MotifSolver(Graph &g, int k, std::vector<uint64_t> &accum, int, int) {
  assert(k >= 3);
  size_t memsize = print_device_info(0);
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
            << " GB, graph_mem = " << mem_graph/1024/1024/1024 << " GB\n";
  //if (memsize < mem_graph) { std::cout << "Memory allocation failed\n"; exit(0); }
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  GraphGPU gg(g);
  gg.init_edgelist(g);
  size_t npatterns = accum.size();
  AccType *h_accumulators = (AccType *)malloc(sizeof(AccType) * npatterns);
  for (int i = 0; i < npatterns; i++) h_accumulators[i] = 0;
  AccType *d_accumulators;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_accumulators, sizeof(AccType) * npatterns));
  clear<<<1, npatterns>>>(d_accumulators);
 
  size_t nwarps = WARPS_PER_BLOCK;
  size_t n_lists = 2;
  if (k == 3) n_lists = 0;
  size_t per_block_vlist_size = nwarps * n_lists * size_t(md) * sizeof(vidType);
  size_t nthreads = BLOCK_SIZE;
  size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  if (k > 3) {
    size_t nb = (memsize*0.9 - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;
  }
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM;
  if (k == 3)
    max_blocks_per_SM = maximum_residency(motif3_warp_edge, nthreads, 0);
  else {
#ifdef FISSION
    max_blocks_per_SM = maximum_residency(star3_warp_edge, nthreads, 0);
    std::cout << "3-star: max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    max_blocks_per_SM = maximum_residency(path4_warp_edge, nthreads, 0);
    std::cout << "4-path: max_blocks_per_SM = " << max_blocks_per_SM << "\n";
#else
    max_blocks_per_SM = maximum_residency(motif4_warp_edge, nthreads, 0);
#endif
  } 
  std::cout << k << "-motif: max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks);
  std::cout << "CUDA " << k << "-motif listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
  size_t list_size = nblocks * per_block_vlist_size;
  std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

  Timer t;
  t.Start();
  if (k == 3) {
    motif3_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    //triangle_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    //wedge_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
  } else if (k == 4) {
#ifdef FISSION
    star3_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    path4_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    cycle4_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    motif4_triangle<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
    //motif4_wedge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
#else
    motif4_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_accumulators);
#endif
  } else {
    std::cout << "Not supported right now\n";
  }
  CUDA_SAFE_CALL(hipMemcpy(h_accumulators, d_accumulators, sizeof(AccType) * npatterns, hipMemcpyDeviceToHost));
  for (size_t i = 0; i < npatterns; i ++) accum[i] = h_accumulators[i];
  t.Stop();

  std::cout << "runtime [cuda_base] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipFree(d_accumulators));
}

