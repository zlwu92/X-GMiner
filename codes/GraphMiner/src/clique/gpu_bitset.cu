#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "bitsets.h"
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

#define BLK_SZ BLOCK_SIZE
typedef hipcub::BlockReduce<AccType, BLK_SZ> BlockReduce;
#include "clique4_warp_edge_bitset.cuh"
#include "clique5_warp_edge_bitset.cuh"
#include "clique6_warp_edge.cuh"
#include "clique7_warp_edge.cuh"
#include "clique8_warp_edge.cuh"

void CliqueSolver(Graph &g, int k, uint64_t &total, int, int) {
  assert(k > 3);
  size_t memsize = print_device_info(0);
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  GraphGPU gg(g);
  gg.init_edgelist(g);
  size_t nwarps = WARPS_PER_BLOCK;
  size_t nthreads = BLK_SZ;
  size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
  size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
  if (nblocks > 65536) nblocks = 65536;
  size_t nb = (memsize - mem_graph) / per_block_vlist_size;
  if (nb < nblocks) nblocks = nb;

  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(clique4_warp_edge_bitset, nthreads, 0);
  if (k==5) max_blocks_per_SM = maximum_residency(clique5_warp_edge_bitset, nthreads, 0);
  if (k==6) max_blocks_per_SM = maximum_residency(clique6_warp_edge, nthreads, 0);
  if (k==7) max_blocks_per_SM = maximum_residency(clique7_warp_edge, nthreads, 0);
  if (k==8) max_blocks_per_SM = maximum_residency(clique8_warp_edge, nthreads, 0);
  std::cout << k << "-clique max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA " << k << "-clique listing/counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
  size_t list_size = nblocks * per_block_vlist_size;
  std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

  MultiBitsets<> adj_lists(nblocks*nwarps, md, md);	
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
  if (k == 4) {
    clique4_warp_edge_bitset<<<nblocks, nthreads>>>(ne, gg, frontier_list, adj_lists, md, d_total);
  } else if (k == 5) {
    clique5_warp_edge_bitset<<<nblocks, nthreads>>>(ne, gg, frontier_list, adj_lists, md, d_total);
  } else if (k == 6) {
    clique6_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
  } else if (k == 7) {
    clique7_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
  } else if (k == 8) {
    clique8_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
  } else {
    std::cout << "Not supported right now\n";
  }
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [gpu] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

