#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "edgelist.h"
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"
#define MAX_PATTERN_SIZE 8
#define BLK_SZ BLOCK_SIZE

typedef hipcub::BlockReduce<AccType, BLK_SZ> BlockReduce;
#include "edge_warp_iterative.cuh"
#include "vertex_warp_iterative.cuh"

void CliqueSolver(Graph &g, int k, uint64_t &total, int, int) {
  assert(k >= 3);
  vidType nv = g.num_vertices();
  eidType ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t memsize = print_device_info(0);
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

  GraphGPU gg(g);
  gg.init_edgelist(g);
  size_t nwarps = WARPS_PER_BLOCK;
  size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
  size_t nthreads = BLK_SZ;
  size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
  if (nblocks > 65536) nblocks = 65536;
  size_t nb = (memsize - mem_graph) / per_block_vlist_size;
  if (nb < nblocks) nblocks = nb;

  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_edge_iterative, nthreads, 0);
  std::cout << k << "-clique max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(16*max_blocks, nblocks);
  std::cout << "CUDA " << k << "-clique listing/counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
  size_t list_size = nblocks * per_block_vlist_size;
  std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
  vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
  CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));

  Timer t;
  t.Start();
  warp_edge_iterative<<<nblocks, nthreads>>>(ne, k, gg, frontier_list, md, d_total);
  //warp_vertex<<<nblocks, nthreads>>>(nv, k, gg, frontier_list, md, d_total);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [cuda_base] = " << t.Seconds() << " sec\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

