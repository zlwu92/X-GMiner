#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "timer.h"
//#include "edgelist.h"
#include "graph_gpu.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bin_search.cuh"

//#define USE_CTA
void TCSolver(Graph &g, uint64_t &total, int, int) {
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";

  GraphGPU gg(g);
  auto nnz = gg.init_edgelist(g);
  std::cout << "Edge parallel: edgelist size = " << nnz << "\n";
  size_t nthreads = BLOCK_SIZE;
#ifdef USE_CTA
  size_t nblocks = ne;
#else
  size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
#endif
  if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_edge, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  //size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  //nblocks = std::min(max_blocks, nblocks);
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
 
  AccType h_total = 0, *d_total;
  CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
  CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
  CUDA_SAFE_CALL(hipDeviceSynchronize());

  Timer t;
  t.Start();
#ifdef USE_CTA
  cta_edge<<<nblocks, nthreads>>>(ne, gg, d_total);
#else
  warp_edge<<<nblocks, nthreads>>>(ne, gg, d_total);
#endif
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  t.Stop();

  std::cout << "runtime [gpu_bs] = " << t.Seconds() << " sec\n";
  std::cout << "throughput = " << double(nnz) / t.Seconds() / 1e9 << " billion Traversed Edges Per Second (TEPS)\n";
  CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
  total = h_total;
  CUDA_SAFE_CALL(hipFree(d_total));
}

