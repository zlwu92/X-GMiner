#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
//#include "edgelist.h"
#include "graph_gpu.h"
#include "graph_partition.h"
#include "scheduler.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_edge.cuh"
#include "bs_warp_vertex.cuh"
#include <thread>

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  int ndevices = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  //eidType nnz = g.init_edgelist();
  //std::cout << "Total edgelist size = " << nnz << "\n";

  if (ndevices < n_gpus) {
    std::cout << "Only " << ndevices << " GPUs available\n";
  } else ndevices = n_gpus;
  PartitionedGraph pg(&g, ndevices);
  pg.edgecut_induced_partition1D();
 
  std::vector<GraphGPU> d_graphs(ndevices);
  Timer t;
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    d_graphs[i].init(*pg.get_subgraph(i), i, ndevices);
    //d_graphs[i].copy_edgelist_to_device(num_tasks, src_ptrs, dst_ptrs);
  }
  t.Stop();
  std::cout << "Total GPU copy time (graph+edgelist) = " << t.Seconds() <<  " sec\n";

  size_t nthreads = BLOCK_SIZE;
  std::vector<AccType> h_counts(ndevices, 0);
  size_t nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_vertex, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks); 
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::vector<AccType *> d_count(ndevices);
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMalloc(&d_count[i], sizeof(AccType)));
  }
  std::vector<std::thread> threads;
  std::vector<Timer> subt(ndevices);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
 
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    threads.push_back(std::thread([&,i]() {
    hipSetDevice(i);
    subt[i].Start();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice));
    //hipMemcpyAsync(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice);
    auto begin = pg.get_local_begin(i);
    auto end = pg.get_local_end(i);
    warp_vertex<<<nblocks, nthreads>>>(begin, end, d_graphs[i], d_count[i]);
    CUDA_SAFE_CALL(hipMemcpy(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost));
    //hipMemcpyAsync(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    subt[i].Stop();
    }));
  }
  for (auto &thread: threads) thread.join();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  for (int i = 0; i < ndevices; i++) total += h_counts[i];
  t.Stop();
  for (int i = 0; i < ndevices; i++)
    std::cout << "runtime[gpu" << i << "] = " << subt[i].Seconds() <<  " sec\n";
  std::cout << "runtime = " << t.Seconds() <<  " sec\n";
}

