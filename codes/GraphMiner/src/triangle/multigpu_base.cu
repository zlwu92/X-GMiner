#include "hip/hip_runtime.h"
// Copyright (c) 2020 MIT
// Author: Xuhao Chen
#include <hipcub/hipcub.hpp>
#include "timer.h"
#include "edgelist.h"
#include "graph_gpu.h"
#include "scheduler.h"
#include "operations.cuh"
#include "cuda_launch_config.hpp"
//#define EVEN_SPLIT

typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;
#include "bs_warp_edge.cuh"
#include <thread>

void TCSolver(Graph &g, uint64_t &total, int n_gpus, int chunk_size) {
  int ndevices = 0;
  eidType nnz = 0;
  CUDA_SAFE_CALL(hipGetDeviceCount(&ndevices));
  size_t memsize = print_device_info(0);
  auto nv = g.num_vertices();
  auto ne = g.num_edges();
  auto md = g.get_max_degree();
  nnz = g.init_edgelist();
  size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
  std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
  std::cout << "Total edgelist size = " << nnz << "\n";

  if (ndevices < n_gpus) {
    std::cout << "Only " << ndevices << " GPUs available\n";
  } else ndevices = n_gpus;

  // split the edgelist onto multiple gpus
#ifdef EVEN_SPLIT
  eidType n_tasks_per_gpu = eidType(nnz-1) / eidType(ndevices) + 1;
  std::vector<eidType> num_tasks(ndevices, n_tasks_per_gpu);
  num_tasks[ndevices-1] = nnz - (ndevices-1) * n_tasks_per_gpu;
#else
  std::vector<vidType*> src_ptrs, dst_ptrs;
  Scheduler scheduler;
  //auto num_tasks = scheduler.split_edgelist(ndevices, g, src_ptrs, dst_ptrs, chunk_size);
  auto num_tasks = scheduler.round_robin(ndevices, g, src_ptrs, dst_ptrs, chunk_size);
#endif

  std::vector<GraphGPU> d_graphs(ndevices);
  Timer t;
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    d_graphs[i].init(g, i, ndevices);
#ifdef EVEN_SPLIT
    d_graphs[i].copy_edgelist_to_device(nnz, g);
#else
    d_graphs[i].copy_edgelist_to_device(num_tasks, src_ptrs, dst_ptrs);
#endif
  }
  t.Stop();
  std::cout << "Total GPU copy time (graph+edgelist) = " << t.Seconds() <<  " sec\n";

  size_t nthreads = BLOCK_SIZE;
  std::vector<AccType> h_counts(ndevices, 0);
  size_t nblocks = 65536; //(n_tasks_per_gpu-1)/WARPS_PER_BLOCK+1;
  //if (nblocks > 65536) nblocks = 65536;
  hipDeviceProp_t deviceProp;
  CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
  int max_blocks_per_SM = maximum_residency(warp_edge, nthreads, 0);
  std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
  size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
  nblocks = std::min(6*max_blocks, nblocks); 
  std::cout << "CUDA triangle counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";

  std::vector<AccType *> d_count(ndevices);
  for (int i = 0; i < ndevices; i++) {
    CUDA_SAFE_CALL(hipSetDevice(i));
    CUDA_SAFE_CALL(hipMalloc(&d_count[i], sizeof(AccType)));
  }
  std::vector<std::thread> threads;
  std::vector<Timer> subt(ndevices);
  CUDA_SAFE_CALL(hipDeviceSynchronize());
 
  t.Start();
  for (int i = 0; i < ndevices; i++) {
    threads.push_back(std::thread([&,i]() {
    hipSetDevice(i);
    subt[i].Start();
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    CUDA_SAFE_CALL(hipMemcpy(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice));
    //hipMemcpyAsync(d_count[i], &h_counts[i], sizeof(AccType), hipMemcpyHostToDevice);
    warp_edge<<<nblocks, nthreads>>>(num_tasks[i], d_graphs[i], d_count[i]);
    CUDA_SAFE_CALL(hipMemcpy(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost));
    //hipMemcpyAsync(&h_counts[i], d_count[i], sizeof(AccType), hipMemcpyDeviceToHost);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    subt[i].Stop();
    }));
  }
  for (auto &thread: threads) thread.join();
  CUDA_SAFE_CALL(hipDeviceSynchronize());
  for (int i = 0; i < ndevices; i++) total += h_counts[i];
  t.Stop();
  for (int i = 0; i < ndevices; i++)
    std::cout << "runtime[gpu" << i << "] = " << subt[i].Seconds() <<  " sec\n";
  std::cout << "runtime = " << t.Seconds() <<  " sec\n";
}

