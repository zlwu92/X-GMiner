#include "hip/hip_runtime.h"
#include "../../include/glumin.h"
#include <hipcub/hipcub.hpp>

#include "../../include/graph_v2_gpu.h"
#include "../../include/glumin/operations.cuh"
#include "../../include/cuda_utils/cuda_launch_config.hpp"
// #include "../../include/glumin/codegen_LUT.cuh"
// #include "../../include/glumin/codegen_utils.cuh"
#include "../../include/glumin/timer.h"
#include "../../include/glumin/binary_encode.h"
#define FISSION
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

#define BLK_SZ BLOCK_SIZE
#include "../../include/glumin/expand_LUT.cuh"

#include "gpu_GF_LUT_kernels/GF_kernels.cuh"
#include "gpu_GF_LUT_kernels/P1_GF_LUT.cuh"
#include "gpu_GF_LUT_kernels/P2_GF_LUT.cuh"
#include "gpu_GF_LUT_kernels/P3_GF_LUT.cuh"
#include "gpu_GF_LUT_kernels/P3_GF_LUT_edge.cuh"
#include "gpu_GF_LUT_kernels/P7_GF_LUT.cuh"
#include "gpu_GF_LUT_kernels/clique5_warp_edge_taskallocate.cuh"
#include "gpu_GF_LUT_kernels/clique5_GF.cuh"

#if 1
__global__ __forceinline__ void clear(AccType *accumulators) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    accumulators[i] = 0;
}

void GLUMIN::PatternSolver_on_GraphFold() {
    LOG_INFO("PatternSolver_on_GraphFold");
    int k = k_num;
    assert(k >= 1);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
    std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
                << " GB, graph_mem = " << mem_graph/1024/1024/1024 << " GB\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";
    // CUDA_SAFE_CALL(hipSetDevice(CUDA_SELECT_DEVICE));
    GraphGPU gg(g);
    gg.init_edgelist(g);

    size_t npatterns = 2;
    AccType *h_counts = (AccType *)malloc(sizeof(AccType) * npatterns);
    for (int i = 0; i < npatterns; i++) h_counts[i] = 0;
    AccType *d_counts;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_counts, sizeof(AccType) * npatterns));
    clear<<<1, npatterns>>>(d_counts);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    
    size_t nwarps = WARPS_PER_BLOCK;
    size_t n_lists = 8;
    size_t n_bitmaps = 2;

    size_t per_block_vlist_size = nwarps * n_lists * size_t(md) * sizeof(vidType);
    size_t per_block_bitmap_size = nwarps * n_bitmaps * ((size_t(md) + BITMAP_WIDTH-1)/BITMAP_WIDTH) * sizeof(vidType);

    size_t nthreads = BLOCK_SIZE;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize*0.9 - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));

    nblocks = 640;
    std::cout << "CUDA pattern listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    size_t bitmap_size = nblocks * per_block_bitmap_size;
    std::cout << "lut rows size: " << bitmap_size/(1024*1024) << " MB\n";
    bitmapType *frontier_bitmap; // each thread has lut rows to store midresult of lut compute
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_bitmap, bitmap_size));

    Roaring_LUTManager<> lut_manager(nblocks * nwarps, WARP_LIMIT, WARP_LIMIT); 

    AccType *G_INDEX, *G_INDEX1, *G_INDEX2, *G_INDEX3;
    AccType nowindex = nblocks * nwarps;
    AccType nowindex1 = nblocks;
    AccType nowindex3 = 500;
    CUDA_SAFE_CALL(hipMalloc((void**) &(G_INDEX), sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(G_INDEX, &nowindex, sizeof(AccType), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMalloc((void**) &(G_INDEX2), sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(G_INDEX2, &nowindex, sizeof(AccType), hipMemcpyHostToDevice));  
    CUDA_SAFE_CALL(hipMalloc((void**) &(G_INDEX1), sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(G_INDEX1, &nowindex1, sizeof(AccType), hipMemcpyHostToDevice));  
    CUDA_SAFE_CALL(hipMalloc((void**) &(G_INDEX3), sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(G_INDEX3, &nowindex3, sizeof(AccType), hipMemcpyHostToDevice));  

    Timer t;
    t.Start();
    // LUT vertex
    if (k == 1) {
        std::cout << "P1 GraphFold LUT\n";
        P1_GF_LUT_warp<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX, lut_manager);
        lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT);
        P1_GF_LUT_block<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX1, lut_manager);
        lut_manager.recreate(1, md, md);
        nblocks = BLOCK_GROUP;
        for (vidType i = 0; i < nv; i++) {
        if (g.get_degree(i) > BLOCK_LIMIT) {
            lut_manager.update_para(1, g.get_degree(i), g.get_degree(i));
            clear_counterlist<<<nblocks, nthreads>>>(gg, frontier_list, md, i);
            GF_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, i);
            P1_GF_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, i);
        }
        }
    }
    // BS Edge
    else if (k == 2){
        std::cout << "P1 GraphFold\n";
        P1_frequency_count<<<nblocks, nthreads>>>(nv, gg, frontier_list, md, d_counts, G_INDEX);
        P1_count_correction<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_counts, G_INDEX2);
    }
    else if (k == 10) {
        std::cout << "P10 GraphFold LUT\n";
        P3_GF_LUT_warp_edge<<<nblocks, nthreads>>>(0, ne, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX, lut_manager);
        lut_manager.recreate(500, md, md);
        P3_GF_LUT_block_edge<<<500, nthreads>>>(0, ne, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX3, lut_manager);
    }
        else if (k == 11) {
        std::cout << "P10 GraphFold\n";
        P3_fused_matching<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_counts, G_INDEX);
    }
    else if (k == 13) {
        std::cout << "P13 GraphFold LUT\n";
        P7_GF_LUT_warp<<<nblocks, nthreads>>>(0, ne, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX, lut_manager);
        lut_manager.recreate(500, md, md);
        P7_GF_LUT_block<<<500, nthreads>>>(0, ne, gg, frontier_list, frontier_bitmap, md, d_counts, G_INDEX3, lut_manager);
    }
    else if (k == 14) {
        std::cout << "P13 GraphFold\n";
        P7_fused_matching<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_counts, G_INDEX);
    }
    else {
        LOG_ERROR("Not supported for P" + std::to_string(k) + " on GraphFold.");
    }
    CUDA_SAFE_CALL(hipMemcpy(h_counts, d_counts, sizeof(AccType) * npatterns, hipMemcpyDeviceToHost));
    // for (size_t i = 0; i < npatterns; i ++) total[i] = h_counts[i];
    total[0] = h_counts[0] - h_counts[1];
    // accum[0] = h_counts[1];
    t.Stop();

    if (k == 2 || k == 11 || k == 14) std::cout << "runtime [GraphFold] = " << t.Seconds() << " sec\n";
    else std::cout << "runtime [GraphFold + LUT] = " << t.Seconds() << " sec\n";
    CUDA_SAFE_CALL(hipFree(d_counts));
}

void GLUMIN::CliqueSolver_on_GraphFold() {
    LOG_INFO("CliqueSolver_on_GraphFold");
    int k = k_num;
    // assert(k > 3);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
    std::cout << "GPU_total_mem = " << memsize << " graph_mem = " << mem_graph << "\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

    // CUDA_SAFE_CALL(hipSetDevice(CUDA_SELECT_DEVICE));
    GraphGPU gg(g);
    gg.init_edgelist(g);
    size_t nwarps = WARPS_PER_BLOCK;
    size_t nthreads = BLK_SZ;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    size_t per_block_vlist_size = nwarps * size_t(4) * size_t(md) * sizeof(vidType);
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;

    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM = maximum_residency(clique5_warp_edge_taskallocate, nthreads, 0);
    double clock_rate = deviceProp.clockRate;

    std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;

    nblocks = std::min(max_blocks, nblocks);
    // nblocks = 640;
    
    std::cout << "CUDA clique listing/counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    AccType h_total = 0, *d_total;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));

    int h_allocator, *d_allocator; // Set 0 for No pre allocate, atomicAdd from zero
    h_allocator = nblocks * nwarps;

    CUDA_SAFE_CALL(hipMalloc((void **)&d_allocator, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpy(d_allocator, &h_allocator, sizeof(int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    BinaryEncode<> sub_graph(nblocks * nwarps, md, md);
    // printf("Sum Warp: %d!!!\n", nblocks * nwarps);

    Timer t;
    t.Start();
    if (k == 5) {
        std::cout << "P5 GraphFold + LUT\n";
        clique5_warp_edge_taskallocate<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total, sub_graph, d_allocator);
    }
    else if (k == 6) {
        std::cout << "P5 GraphFold\n";
        clique5_GF<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total, sub_graph, d_allocator);
    }
    else {
        LOG_ERROR("Not supported for P" + std::to_string(k) + " on GraphFold.");
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    t.Stop();

    if (k == 6) std::cout << "runtime [GraphFold] = " << t.Seconds() << " sec\n";
    else std::cout << "runtime [GraphFold + LUT] = " << t.Seconds() << " sec\n";
    CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
    total[0] = h_total;
    CUDA_SAFE_CALL(hipFree(d_total));
    CUDA_SAFE_CALL(hipFree(d_allocator));
}

#endif