#include "hip/hip_runtime.h"
#include "../../include/glumin.h"
#include <hipcub/hipcub.hpp>

#include "../../include/graph_v2_gpu.h"
#include "../../include/glumin/operations.cuh"
#include "../../include/cuda_utils/cuda_launch_config.hpp"
#include "../../include/glumin/codegen_LUT.cuh"
#include "../../include/glumin/codegen_utils.cuh"
#include "../../include/glumin/timer.h"
#include "../../include/glumin/binary_encode.h"
#define FISSION
typedef hipcub::BlockReduce<AccType, BLOCK_SIZE> BlockReduce;

#include "GM_LUT.cuh"
#include "GM_build_LUT.cuh"
#include "GM_LUT_deep.cuh"
#include "GM_BS_vertex.cuh"
#include "GM_BS_edge.cuh"

#define BLK_SZ BLOCK_SIZE
#include "GM_kernels/clique4_warp_edge.cuh"
#include "GM_kernels/clique5_warp_edge.cuh"
#include "GM_kernels/clique6_warp_edge.cuh"
#include "GM_kernels/clique7_warp_edge.cuh"


#include "GM_LUT_kernels/clique4_warp_vertex_subgraph.cuh"
#include "GM_LUT_kernels/clique5_warp_edge_subgraph.cuh"
#include "GM_LUT_kernels/clique6_warp_edge_subgraph.cuh"
#include "GM_LUT_kernels/clique7_warp_edge_subgraph.cuh"

#include "GM_LUT_kernels/P2_profile.cuh"
#include "GM_LUT_kernels/P3_profile.cuh"
#include "GM_LUT_kernels/P1_profile.cuh"
#include "GM_LUT_kernels/P6_profile.cuh"
#include "GM_kernels/P2_profile.cuh"

#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/fill.h>
#include <thrust/execution_policy.h>

__global__ __forceinline__ void clear(AccType *accumulators) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    accumulators[i] = 0;
}


void GLUMIN::PatternSolver_on_G2Miner(Graph_V2& g) {
    LOG_INFO("PatternSolver_on_G2Miner without LUT");
    // Graph_V2 g(prefix, use_dag);
    // g = Graph_V2(prefix, use_dag);
    // g.print_meta_data();
    // g = std::make_unique<Graph_V2>(prefix, use_dag);
    
    int k = local_patternId;
    assert(k >= 1);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
    std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
              << " GB, graph_mem = " << mem_graph/1024/1024 << " MB\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";
    // CUDA_SAFE_CALL(hipSetDevice(CUDA_SELECT_DEVICE));
    GraphGPU gg(g);
    gg.init_edgelist(g);

#if 1
    size_t npatterns = 3;
    AccType *h_counts = (AccType *)malloc(sizeof(AccType) * npatterns);
    for (int i = 0; i < npatterns; i++) h_counts[i] = 0;
    AccType *d_counts;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_counts, sizeof(AccType) * npatterns));
    clear<<<1, npatterns>>>(d_counts);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
  
    size_t nwarps = WARPS_PER_BLOCK;
    size_t n_lists;
    size_t n_bitmaps;

    n_lists = 5;
    n_bitmaps = 1;

    vidType switch_lut = 1;
    switch_lut = Select_func(nv, ne, md);

    size_t per_block_vlist_size = nwarps * n_lists * size_t(md) * sizeof(vidType);
    size_t per_block_bitmap_size = nwarps * n_bitmaps * ((size_t(md) + BITMAP_WIDTH-1)/BITMAP_WIDTH) * sizeof(vidType);

    size_t nthreads = BLOCK_SIZE;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize*0.9 - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM;
    if (k == 1){
      max_blocks_per_SM = maximum_residency(GM_LUT_warp, nthreads, 0);
      std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    }
    else {
      max_blocks_per_SM = maximum_residency(GM_LUT_warp, nthreads, 0);
      std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    } 
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;

    nblocks = std::min(6*max_blocks, nblocks);

    nblocks = 640;
    std::cout << "CUDA pattern listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    size_t bitmap_size = nblocks * per_block_bitmap_size;
    std::cout << "lut rows size: " << bitmap_size/(1024*1024) << " MB\n";
    bitmapType *frontier_bitmap; // each thread has lut rows to store midresult of lut compute
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_bitmap, bitmap_size));

    LUTManager<> lut_manager(nblocks * nwarps, WARP_LIMIT, WARP_LIMIT, true); 
    std::cout << "nblocks*nwarps = " << nblocks * nwarps << " WARP_LIMIT = " << WARP_LIMIT << "\n";
    // split vertex tasks
    std::vector<vidType> vid_warp, vid_block, vid_global;

    for (int vid = 0; vid < nv; ++vid) {
      auto degree = g.get_degree(vid);
      if (degree <= WARP_LIMIT) {
          vid_warp.push_back(vid);
      } else if (degree <= BLOCK_LIMIT) {
          vid_block.push_back(vid);
      } else {
          vid_global.push_back(vid);
      }
    }
    vidType vid_warp_size = vid_warp.size();
    vidType vid_block_size = vid_block.size();
    vidType vid_global_size = vid_global.size();
    // std::cout << "warp_task: " << vid_warp_size << " block_task: " << vid_block_size << " global_task: " << vid_global_size << "\n";
    vidType *d_vid_warp;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_vid_warp, vid_warp_size * sizeof(vidType)));
    CUDA_SAFE_CALL(hipMemcpy(d_vid_warp, vid_warp.data(), vid_warp_size * sizeof(vidType), hipMemcpyHostToDevice));

    vidType *d_vid_block;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_vid_block, vid_block_size * sizeof(vidType)));
    CUDA_SAFE_CALL(hipMemcpy(d_vid_block, vid_block.data(), vid_block_size * sizeof(vidType), hipMemcpyHostToDevice));

    Timer t;
    t.Start();
    // G2Miner
    if (k == 1){
      std::cout << "P1 Run G2Miner\n";
      P1_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 2){
      std::cout << "P2 Run G2Miner\n";
      P2_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 3){
      std::cout << "P3 Run G2Miner\n";
      std::cout << "nblocks = " << nblocks << " nthreads = " << nthreads << " ne = " << ne << " md = " << md << "\n";
      P3_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 6){
      std::cout << "P6 Run G2Miner\n";
      P6_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 7){
      std::cout << "P7 Run G2Miner\n";
      P7_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 7){
      std::cout << "P7 Run G2Miner\n";
      P7_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 8){
      std::cout << "P8 Run G2Miner\n";
      P8_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 9){
      std::cout << "P9 Run G2Miner\n";
      P9_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 10){
      std::cout << "P10 Run G2Miner\n";
      P10_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 11){
      std::cout << "P11 Run G2Miner\n";
      P11_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 12){
      std::cout << "P12 Run G2Miner\n";
      P12_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 13){
      std::cout << "P13 Run G2Miner\n";
      P13_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 14){
      std::cout << "P14 Run G2Miner\n";
      P14_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 15){
      std::cout << "P15 Run G2Miner\n";
      P15_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 16){
      std::cout << "P16 Run G2Miner\n";
      P16_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 17){
      std::cout << "P17 Run G2Miner\n";
      P17_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }              
    else if (k == 18){
      std::cout << "P18 Run G2Miner\n";
      P18_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 19){
      std::cout << "P19 Run G2Miner\n";
      P19_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 20){
      std::cout << "P20 Run G2Miner\n";
      P20_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 21){
      std::cout << "P21 Run G2Miner\n";
      P21_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else if (k == 22){
      std::cout << "P22 Run G2Miner\n";
      P22_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
    }
    else {
      LOG_ERROR("Unsupported pattern: " + std::to_string(k) + " for G2Miner without LUT.");
    }
    CUDA_SAFE_CALL(hipMemcpy(h_counts, d_counts, sizeof(AccType) * npatterns, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < npatterns; i ++) total[i] = h_counts[i];
    t.Stop();

    total_time[0] += t.Seconds();
    std::cout << "runtime [G2Miner] = " << t.Seconds() << " sec\n";
    CUDA_SAFE_CALL(hipFree(d_counts));
#endif
}

#if 1
void GLUMIN::PatternSolver_LUT_on_G2Miner(Graph_V2& g) {
    LOG_INFO("PatternSolver_on_G2Miner with LUT");
    int k = local_patternId;
    assert(k >= 1);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
    std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
              << " GB, graph_mem = " << mem_graph/1024/1024 << " MB\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";
    // CUDA_SAFE_CALL(hipSetDevice(CUDA_SELECT_DEVICE));
    GraphGPU gg(g);
    gg.init_edgelist(g);

    size_t npatterns = 3;
    AccType *h_counts = (AccType *)malloc(sizeof(AccType) * npatterns);
    for (int i = 0; i < npatterns; i++) h_counts[i] = 0;
    AccType *d_counts;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_counts, sizeof(AccType) * npatterns));
    clear<<<1, npatterns>>>(d_counts);
    CUDA_SAFE_CALL(hipDeviceSynchronize());
  
    size_t nwarps = WARPS_PER_BLOCK;
    size_t n_lists;
    size_t n_bitmaps;
    n_lists = 7;
    n_bitmaps = 3;

    vidType switch_lut = use_lut;//1;
    // switch_lut = Select_func(nv, ne, md);
    std::cout << "switch_lut = " << switch_lut << "\n";
    size_t per_block_vlist_size = nwarps * n_lists * size_t(md) * sizeof(vidType);
    size_t per_block_bitmap_size = nwarps * n_bitmaps * ((size_t(md) + BITMAP_WIDTH-1)/BITMAP_WIDTH) * sizeof(vidType);

    size_t nthreads = BLOCK_SIZE;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize*0.9 - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;
    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM;
    if (k == 1){
      max_blocks_per_SM = maximum_residency(GM_LUT_warp, nthreads, 0);
      std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    }
    else {
      max_blocks_per_SM = maximum_residency(GM_LUT_warp, nthreads, 0);
      std::cout << "max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    } 
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;
    std::cout << "max_blocks = " << max_blocks << "\n";
    nblocks = std::min(6*max_blocks, nblocks);

    nblocks = 640;
    std::cout << "CUDA pattern listing (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    size_t bitmap_size = nblocks * per_block_bitmap_size;
    std::cout << "lut rows size: " << bitmap_size/(1024*1024) << " MB\n";
    bitmapType *frontier_bitmap; // each thread has lut rows to store midresult of lut compute
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_bitmap, bitmap_size));

    LUTManager<> lut_manager(nblocks * nwarps, WARP_LIMIT, WARP_LIMIT, true); 
    std::cout << "WARP_LIMIT: " << WARP_LIMIT << ", BLOCK_LIMIT: " << BLOCK_LIMIT << "\n";
    // split vertex tasks
    std::vector<vidType> vid_warp, vid_block, vid_global;

    for (int vid = 0; vid < nv; ++vid) {
      auto degree = g.get_degree(vid);
      if (degree <= WARP_LIMIT) {
          vid_warp.push_back(vid);
      } else if (degree <= BLOCK_LIMIT) {
          vid_block.push_back(vid);
      } else {
          vid_global.push_back(vid);
      }
    }
    vidType vid_warp_size = vid_warp.size();
    vidType vid_block_size = vid_block.size();
    vidType vid_global_size = vid_global.size();
    std::cout << "warp_task: " << vid_warp_size << " block_task: " << vid_block_size << " global_task: " << vid_global_size << "\n";
    vidType *d_vid_warp;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_vid_warp, vid_warp_size * sizeof(vidType)));
    CUDA_SAFE_CALL(hipMemcpy(d_vid_warp, vid_warp.data(), vid_warp_size * sizeof(vidType), hipMemcpyHostToDevice));

    vidType *d_vid_block;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_vid_block, vid_block_size * sizeof(vidType)));
    CUDA_SAFE_CALL(hipMemcpy(d_vid_block, vid_block.data(), vid_block_size * sizeof(vidType), hipMemcpyHostToDevice));

    std::ofstream out("/home/zlwu/workspace/2-graphmining/X-GMiner/results/g2miner_glumin_memory_profiling.csv", std::ios::app);
    // out << nv << "," << ne << "," << md << ",";
    // size_t graphsize = (nv+1)*sizeof(eidType) + 2*ne*sizeof(vidType);
    // out << (double)graphsize / 1024.0 / 1024.0 << ",";
    // out << (double)list_size / 1024.0 / 1024.0 + (double)bitmap_size / 1024.0 / 1024.0 + 
    //       (double)lut_manager.max_LUT_size_ * lut_manager.LUT_num_ * sizeof(vidType) / 1024.0 / 1024.0 << ",";
    // return;

    // vidType* d_work_depth_each_warp;
    // int num_warps = std::min((vidType)nblocks, vid_block_size) * nwarps;//nblocks * WARPS_PER_BLOCK;//
    // CUDA_SAFE_CALL(hipMalloc((void **)&d_work_depth_each_warp, num_warps * sizeof(vidType)));
    // CUDA_SAFE_CALL(hipMemset(d_work_depth_each_warp, 0, num_warps * sizeof(vidType)));
    // std::vector<vidType> work_depth_each_warp(num_warps);

    // ideal case: transform all CSR neighbor list to adjacency bitmap
    // double lut_gpu_mem = (double)(nv + 31) / 32 * nv * sizeof(vidType) / 1024.0 / 1024.0;
    // std::cout << "lut_gpu_mem: " << lut_gpu_mem << " MB\n";

    if (switch_lut) {
      lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
      // lut_manager.recreate(nblocks, md, md, true);
      CUDA_SAFE_CALL(hipDeviceSynchronize());
    }

    std::vector<vidType> workload(nblocks * nthreads, 0);
    std::cout << "workload size:" << nblocks * nthreads << ", " << workload.size() << ", " << workload.capacity() << "\n";
    vidType *d_workload;
    if (prof_workload) {
      CUDA_SAFE_CALL(hipMalloc((void **)&d_workload, nblocks * nthreads * sizeof(vidType)));
      CUDA_SAFE_CALL(hipMemset(d_workload, 0, nblocks * nthreads * sizeof(vidType)));
    }
    else {
      d_workload = nullptr;
    }
    std::vector<vidType> edgecheck(nblocks * nthreads);
    vidType *d_edgecheck, *d_edgecheck2; 
    AccType* d_edgecheck_cnt;
    uint64_t edgecheck_size = 1343125092;
    std::vector<vidType> edgecheck2;//
    if (prof_edgecheck) {
      CUDA_SAFE_CALL(hipMalloc((void **)&d_edgecheck, nblocks * nthreads * sizeof(vidType)));
      CUDA_SAFE_CALL(hipMemset(d_edgecheck, 0, nblocks * nthreads * sizeof(vidType)));

      edgecheck2.resize(edgecheck_size);
      CUDA_SAFE_CALL(hipMallocManaged((void **)&d_edgecheck2, edgecheck_size * sizeof(vidType)));
      CUDA_SAFE_CALL(hipMemset(d_edgecheck2, 0, edgecheck_size * sizeof(vidType)));
      thrust::fill(thrust::device, d_edgecheck2, d_edgecheck2 + edgecheck_size, -1);
      std::cout << "edgecheck2 size: " << edgecheck_size * sizeof(vidType) / 1024.0 / 1024.0 << " MB\n";

      CUDA_SAFE_CALL(hipMalloc((void **)&d_edgecheck_cnt, sizeof(AccType)));
      CUDA_SAFE_CALL(hipMemset(d_edgecheck_cnt, 0, sizeof(AccType)));
    }
    else {
      d_edgecheck = nullptr;
    }

    std::cout << __LINE__ << "length = " << nblocks * nthreads << "\n";
    float time[3];
    Timer t;
    t.Start();
    hipEvent_t e1, e2, e3, e4;
    GPUTimer t0;
    float elapsedTime;
    hipEventCreate(&e1);
    hipEventCreate(&e2);
    hipEventCreate(&e3);
    hipEventCreate(&e4);
    hipEventRecord(e1, 0);
    // G2Miner + LUT
    if (k == 1) {
      std::cout << "P1 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P1_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          // lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          // t0.start();
          if (prof_workload) {
            PRINT_GREEN("P1_GM_LUT_block_workload_test");
            P1_GM_LUT_block_workload_test<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                  frontier_list, frontier_bitmap, md, d_counts, lut_manager, d_workload); 
          }
          else {
            P1_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                  frontier_list, frontier_bitmap, md, d_counts, lut_manager);
          }
          // t0.end_with_sync();
          // time[0] = t0.elapsed() / 1000;
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          // t0.start();
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P1_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
          // t0.end_with_sync();
          // time[2] = t0.elapsed() / 1000;
        }
      }
      else {
        P1_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 2) {
      PRINT_GREEN("P2 G2Miner + LUT");
      if (switch_lut){
        if (WARP_LIMIT != 0) {
          std::cout << __LINE__ << " vid_warp_size: " << vid_warp_size << ", nthreads: " << nthreads << "\n";
          P2_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_block_size) {
          std::cout << __LINE__ << "length = " << nblocks * nthreads << "\n";
          std::cout << __LINE__ << " vid_block_size: " << vid_block_size << ", nthreads: " << nthreads << "\n";
          // lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          // t0.start();
          if (prof_workload) {
            PRINT_GREEN("P2_GM_LUT_block_workload_test");
            P2_GM_LUT_block_workload_test<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                    frontier_list, frontier_bitmap, md, d_counts, lut_manager, 
                                                    d_workload
                                                  );
          } else if (prof_edgecheck) {
            PRINT_GREEN("P2_GM_LUT_block_profile_edgecheck_only");
            // P2_GM_LUT_block_edgecheck_test<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
            //                                         frontier_list, frontier_bitmap, md, d_counts, lut_manager, 
            //                                         d_edgecheck, d_edgecheck2, d_edgecheck_cnt
            //                                       );
            P2_GM_LUT_block_profile_edgecheck_only<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                      frontier_list, frontier_bitmap, md, d_counts, lut_manager, 
                                                      d_edgecheck2, d_edgecheck_cnt
                                                    );
          }
          else {
            P2_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                  frontier_list, frontier_bitmap, md, d_counts, lut_manager);
          }
          // t0.end_with_sync();
          // time[0] = t0.elapsed() / 1000;
        }
        if (vid_global_size){
          std::cout << __LINE__ << " vid_global_size: " << vid_global_size << "\n";
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          // t0.start();
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P2_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
          // t0.end_with_sync();
          // time[2] = t0.elapsed() / 1000;
        }
      
      }
      else {
        if (prof_workload) {
          PRINT_GREEN("P2_GM_workload_test");
          if (use_vert_para) {
            PRINT_GREEN("P2_GM_vert_parallel_test");
            P2_GM_vert_parallel_test<<<nblocks, nthreads>>>(ne, gg, 
                                    frontier_list, frontier_bitmap, md, d_counts, lut_manager, d_workload);
          } else {
            P2_GM_workload_test<<<nblocks, nthreads>>>(ne, gg, 
              frontier_list, frontier_bitmap, md, d_counts, lut_manager, d_workload);
          }
        }
        else {
          P2_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
      }
    }
    else if (k == 3) {
      std::cout << "P3 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P3_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          // lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          // t0.start();
          if (prof_workload) {
            PRINT_GREEN("P3_GM_LUT_block_workload_test");
            P3_GM_LUT_block_workload_test<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                  frontier_list, frontier_bitmap, md, d_counts, lut_manager, d_workload);
          }
          else{
            P3_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                                  frontier_list, frontier_bitmap, md, d_counts, lut_manager);
          }
          // t0.end_with_sync();
          // time[0] = t0.elapsed() / 1000;
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          // t0.start();
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P3_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
          // t0.end_with_sync();
          // time[2] = t0.elapsed() / 1000;
        }
      }
      else {
        P3_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 6) {
      std::cout << "P6 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P6_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          // lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          if (prof_workload) {
            PRINT_GREEN("P6_GM_LUT_block_workload_test");
            P6_GM_LUT_block_workload_test<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                            frontier_list, frontier_bitmap, md, d_counts, lut_manager, d_workload);
          }
          else {
            P6_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, 
                                            frontier_list, frontier_bitmap, md, d_counts, lut_manager);
            }
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P6_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P6_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 7) {
      std::cout << "P7 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P7_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P7_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P7_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P7_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 8) {
      std::cout << "P8 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P8_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P8_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P8_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P8_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 9) {
      std::cout << "P9 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P9_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P9_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P9_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P9_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 10) {
      std::cout << "P10 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P10_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P10_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P10_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P10_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 11) {
      std::cout << "P11 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P11_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P11_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P11_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P11_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 12) {
      std::cout << "P12 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P12_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P12_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P12_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P12_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 13) {
      std::cout << "P13 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P13_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P13_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P13_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P13_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 14) {
      std::cout << "P14 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P14_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P14_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P14_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P14_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 15) {
      std::cout << "P15 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P15_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P15_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P15_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P15_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 16) {
      std::cout << "P16 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P16_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P16_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P16_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P16_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 17) {
      std::cout << "P17 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P17_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P17_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P17_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P17_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 18) {
      std::cout << "P18 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P18_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P18_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P18_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P18_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 19) {
      std::cout << "P19 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P19_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P19_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P19_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P19_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 20) {
      std::cout << "P20 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P20_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P20_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P20_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P20_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 21) {
      std::cout << "P21 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P21_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P21_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P21_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P21_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else if (k == 22) {
      std::cout << "P22 G2Miner + LUT\n";
      if (switch_lut){
        if (WARP_LIMIT != 0) P22_GM_LUT_warp<<<nblocks, nthreads>>>(0, vid_warp_size, d_vid_warp, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        if (vid_block_size) {
          lut_manager.recreate(nblocks, BLOCK_LIMIT, BLOCK_LIMIT, true);
          P22_GM_LUT_block<<<nblocks, nthreads>>>(0, vid_block_size, d_vid_block, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
        }
        if (vid_global_size){
          lut_manager.recreate(1, md, md, true);
          nblocks = BLOCK_GROUP;
          for (vidType i = 0; i < vid_global_size; i++) {
            vidType task_id = vid_global[i];
            lut_manager.update_para(1, g.get_degree(task_id), g.get_degree(task_id), true);
            GM_build_LUT<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
            P22_GM_LUT_global<<<nblocks, nthreads>>>(0, nv, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager, task_id);
          }
        }
      }
      else {
        P22_GM<<<nblocks, nthreads>>>(ne, gg, frontier_list, frontier_bitmap, md, d_counts, lut_manager);
      }
    }
    else {
      std::cout << "Not supported right now\n";
    }
    hipEventRecord(e2, 0);
    hipEventSynchronize(e2);
    hipEventElapsedTime(&elapsedTime, e1, e2);
    t.Stop();
    CUDA_SAFE_CALL(hipMemcpy(h_counts, d_counts, sizeof(AccType) * npatterns, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < npatterns; i ++) total[i] = h_counts[i];

    // CUDA_SAFE_CALL(hipMemcpy(work_depth_each_warp.data(), d_work_depth_each_warp, num_warps * sizeof(vidType), hipMemcpyDeviceToHost));
    out.close();
    out.open("/data-ssd/home/zhenlin/workspace/graphmining/X-GMiner/results/work_depth_per_warp_glumin_g2miner_lut.csv", std::ios::app);
    // out << "P" << k << "_LUT,";
    // for (size_t i = 0; i < work_depth_each_warp.size(); i++) {
    //   out << work_depth_each_warp[i];
    //   if (i < work_depth_each_warp.size() - 1)  out << ",";
    // }
    // out << "\n";
    out.close();
    std::cout << __LINE__ << "length = " << nblocks * nthreads << "\n";
  
    out.open("../results/prof_glumin_LUT_kernel_time_percentage.csv", std::ios::app);
    hipDeviceProp_t prop;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&prop, 0));
    std::string prop_name(prop.name);
    if (switch_lut) {
      if (prop_name.find("3090") != std::string::npos) {
        out << "P" << k << ",3090,";
        out << time[0] << "," << time[2] << "," << t.Seconds() << ",";
      } else if (prop_name.find("6000") != std::string::npos) {
        out << "P" << k << ",ada6000,";
        std::cout << "here!!!\n";
        out << time[0] << "," << time[2] << "," << t.Seconds() << ",";
      }
    } else {
      if (prop_name.find("3090") != std::string::npos) {
        // out << "3090,";
        out << elapsedTime / 1000.0 << "\n";
      } else if (prop_name.find("6000") != std::string::npos) {
        // out << "ada6000,";
        std::cout << "here~~~\n";
        out << elapsedTime / 1000.0 << "\n";
      }
    }

    if (switch_lut) {
      std::cout << "runtime [G2Miner + LUT] = " << t.Seconds() << " sec\n";
      total_time[0] += t.Seconds();
    }
    else {
      std::cout << "runtime [G2Miner] = " << elapsedTime / 1000.0 << " sec\n";
      total_time[0] += elapsedTime / 1000.0;
    }
    out.close();
    
    if (prof_workload) {
      std::string file = "../results/prof_glumin_kernel_workload_" + data_name + ".csv";
      out.open(file);
      std::cout << __LINE__ << "length = " << workload.size() << "\n";
      CUDA_SAFE_CALL(hipMemcpy(workload.data(), d_workload, sizeof(vidType) * workload.size(), hipMemcpyDeviceToHost));
      for (int i = 0; i < workload.size(); i++) {
        out << workload[i] << "\n";
        // if (workload[i] == 0) printf("zero workload!\n");
      }
      out.close();
      unsigned long total = std::accumulate(workload.begin(), workload.end(), 0LL);
      auto max = std::max_element(workload.begin(), workload.end());
      auto min = std::min_element(workload.begin(), workload.end());
      std::cout << workload.end() - workload.begin() << ", " << workload.size() << "\n";
      float max_min = (float) *max / *min;
      out.open("../results/prof_glumin_kernel_workload.csv", std::ios::app);
      std::cout << total << "," << *max << "," << *min << "," << (float) total / workload.size() << "\n";
      out << total << "," << max_min << "," << (float) total / workload.size();
    }
    if (prof_edgecheck) {
      std::string file = "../results/prof_glumin_kernel_edgecheck_" + data_name + ".csv";
      out.open(file);
      CUDA_SAFE_CALL(hipMemcpy(edgecheck.data(), d_edgecheck, sizeof(vidType) * edgecheck.size(), hipMemcpyDeviceToHost));
      for (int i = 0; i < nblocks * nthreads; i++) {
        out << edgecheck[i] << "\n";
      }
      out.close();
      // calculate max of edgecheck
      int max_edgecheck = *std::max_element(edgecheck.begin(), edgecheck.end());
      int total = std::accumulate(edgecheck.begin(), edgecheck.end(), 0);
      std::cout << "max_edgecheck = " << max_edgecheck << "\n";
      std::cout << "max memory = " << 1.0 * max_edgecheck * edgecheck.size() * sizeof(vidType) / 1024 / 1024 / 1024 << " GB\n";
      std::cout << "real memory = " << 1.0 * total * sizeof(vidType) / 1024 / 1024 / 1024 << " GB\n";
    
      AccType edgecheck_cnt;
      CUDA_SAFE_CALL(hipMemcpy(&edgecheck_cnt, d_edgecheck_cnt, sizeof(AccType), hipMemcpyDeviceToHost));
      std::cout << "edgecheck_cnt = " << edgecheck_cnt << "\n";

      CUDA_SAFE_CALL(hipMemcpy(edgecheck2.data(), d_edgecheck2, sizeof(vidType) * edgecheck_cnt, hipMemcpyDeviceToHost));
      out.open("../results/prof_glumin_kernel_edgecheck2_" + data_name + ".csv");
      std::map<std::pair<int, int>, int> pair_counts;
      for (int i = 0; i < edgecheck_cnt; i+=2) {
        // out << edgecheck2[i] << "," << edgecheck2[i+1] << "\n";
        pair_counts[{edgecheck2[i], edgecheck2[i + 1]}]++;
      }
      // 打印每对(pair)出现的次数
      // for (const auto& pair : pair_counts) {
      //   std::cout << "Pair {" << pair.first.first << ", " << pair.first.second << "}: " << pair.second << " times" << std::endl;
      // }
      int unique_pairs = pair_counts.size();
      std::cout << "Total number of unique pairs: " << unique_pairs << std::endl;
      // 统计总次数、最大次数和最小次数
      int total_count = 0;
      int max_count = std::numeric_limits<int>::min();
      int min_count = std::numeric_limits<int>::max();
      
      for (const auto& pair : pair_counts) {
          total_count += pair.second;
          max_count = std::max(max_count, pair.second);
          min_count = std::min(min_count, pair.second);
      }
      std::cout << "sum: " << total_count << ", max: " << max_count << ", min: " << min_count << "\n";
      out.close();
    }

    CUDA_SAFE_CALL(hipFree(d_counts));
}

void GLUMIN::CliqueSolver_LUT_on_G2Miner(Graph_V2& g) {
    LOG_INFO("CliqueSolver_on_G2Miner + LUT");
    int k = k_num;
    assert(k > 3);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
      std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
              << " GB, graph_mem = " << mem_graph/1024/1024 << " MB\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

    GraphGPU gg(g);
    gg.init_edgelist(g);
    size_t nwarps = WARPS_PER_BLOCK;
    size_t nthreads = BLK_SZ;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;

    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM = maximum_residency(clique4_warp_vertex_subgraph, nthreads, 0);
    double clock_rate = deviceProp.clockRate;

    if (k==5) max_blocks_per_SM = maximum_residency(clique5_warp_edge_subgraph, nthreads, 0);
    if (k==6) max_blocks_per_SM = maximum_residency(clique6_warp_edge_subgraph, nthreads, 0);
    if (k==7) max_blocks_per_SM = maximum_residency(clique7_warp_edge_subgraph, nthreads, 0);
    std::cout << k << "-clique max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;

    nblocks = std::min(max_blocks, nblocks);
    
    std::cout << "CUDA " << k << "-clique listing/counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    AccType h_total = 0, *d_total;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipDeviceSynchronize());

    BinaryEncode<> sub_graph(nblocks * nwarps, md, md);
    printf("Sum Warp: %d!!!\n", nblocks * nwarps);

    Timer t;
    t.Start();
    if (k == 4) {
      std::cout << "P4 G2Miner + LUT\n";
      clique4_warp_vertex_subgraph<<<nblocks, nthreads>>>(nv, gg, frontier_list, md, d_total, sub_graph);
    } else if (k == 5) {
      std::cout << "P5 G2Miner + LUT\n";
      clique5_warp_edge_subgraph<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total, sub_graph);
    } else if (k == 6) {
      std::cout << "P23 G2Miner + LUT\n";
      clique6_warp_edge_subgraph<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total, sub_graph);
    } else if (k == 7) {
      std::cout << "P24 G2Miner + LUT\n";
      clique7_warp_edge_subgraph<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total, sub_graph);
    } else {
      LOG_ERROR("Unsupported pattern: " + std::to_string(k) + " for G2Miner with LUT.");
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    t.Stop();

    total_time[0] += t.Seconds();
    std::cout << "runtime [G2Miner + LUT] = " << t.Seconds() << " sec\n";
    CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
    total[0] = h_total;
    CUDA_SAFE_CALL(hipFree(d_total));
}

#endif

#if 1
void GLUMIN::CliqueSolver_on_G2Miner(Graph_V2& g) {
    LOG_INFO("CliqueSolver_on_G2Miner without LUT");
    int k = k_num;
    assert(k > 3);
    size_t memsize = print_device_info(0);
    vidType nv = g.num_vertices();
    eidType ne = g.num_edges();
    auto md = g.get_max_degree();
    size_t mem_graph = size_t(nv+1)*sizeof(eidType) + size_t(2)*size_t(ne)*sizeof(vidType);
    std::cout << "GPU_total_mem = " << memsize/1024/1024/1024
              << " GB, graph_mem = " << mem_graph/1024/1024 << " MB\n";
    if (memsize < mem_graph) std::cout << "Graph too large. Unified Memory (UM) required\n";

    GraphGPU gg(g);
    gg.init_edgelist(g);
    size_t nwarps = WARPS_PER_BLOCK;
    size_t nthreads = BLK_SZ;
    size_t nblocks = (ne-1)/WARPS_PER_BLOCK+1;
    size_t per_block_vlist_size = nwarps * size_t(k-3) * size_t(md) * sizeof(vidType);
    if (nblocks > 65536) nblocks = 65536;
    size_t nb = (memsize - mem_graph) / per_block_vlist_size;
    if (nb < nblocks) nblocks = nb;

    hipDeviceProp_t deviceProp;
    CUDA_SAFE_CALL(hipGetDeviceProperties(&deviceProp, 0));
    int max_blocks_per_SM = maximum_residency(clique4_warp_edge, nthreads, 0);

    if (k==5) max_blocks_per_SM = maximum_residency(clique5_warp_edge, nthreads, 0);  
    if (k==6) max_blocks_per_SM = maximum_residency(clique6_warp_edge, nthreads, 0);
    if (k==7) max_blocks_per_SM = maximum_residency(clique7_warp_edge, nthreads, 0);
    std::cout << k << "-clique max_blocks_per_SM = " << max_blocks_per_SM << "\n";
    size_t max_blocks = max_blocks_per_SM * deviceProp.multiProcessorCount;

    nblocks = std::min(16*max_blocks, nblocks);  
    
    std::cout << "CUDA clique listing/counting (" << nblocks << " CTAs, " << nthreads << " threads/CTA)\n";
    size_t list_size = nblocks * per_block_vlist_size;
    std::cout << "frontier list size: " << list_size/(1024*1024) << " MB\n";
    vidType *frontier_list; // each warp has (k-3) vertex sets; each set has size of max_degree
    CUDA_SAFE_CALL(hipMalloc((void **)&frontier_list, list_size));

    AccType h_total = 0, *d_total;
    CUDA_SAFE_CALL(hipMalloc((void **)&d_total, sizeof(AccType)));
    CUDA_SAFE_CALL(hipMemcpy(d_total, &h_total, sizeof(AccType), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipDeviceSynchronize());


    Timer t;
    t.Start();
    if (k == 4) {
      std::cout << "P4 Run G2Miner\n";
      clique4_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
    } else if (k == 5) {
      std::cout << "P5 Run G2Miner\n";
      clique5_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
    } else if (k == 6) {
      std::cout << "P23 Run G2Miner\n";
      clique6_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
    } else if (k == 7) {
      std::cout << "P24 Run G2Miner\n";
      clique7_warp_edge<<<nblocks, nthreads>>>(ne, gg, frontier_list, md, d_total);
    } else {
      LOG_ERROR("Unsupported pattern: " + std::to_string(k) + " for G2Miner without LUT.");
    }
    CUDA_SAFE_CALL(hipDeviceSynchronize());
    t.Stop();

    total_time[0] = t.Seconds();
    std::cout << "runtime [G2Miner] = " << t.Seconds() << " sec\n";
    CUDA_SAFE_CALL(hipMemcpy(&h_total, d_total, sizeof(AccType), hipMemcpyDeviceToHost));
    total[0] += h_total;
    CUDA_SAFE_CALL(hipFree(d_total));
}
#endif
